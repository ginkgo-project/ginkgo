
#include <hip/hip_runtime.h>
#include <iostream>
constexpr unsigned int default_block_size = 512;


__global__ void parsinv_kernel( 
    int n, // matrix size
    int Lnnz, // number of nonzeros in LT stored in CSR, upper triangular  (equivalent to L in CSC)
    int *Lrowptr, // row pointer L
    int *Lcolidx, //col index L 
    double *Lval, // val array L
    int Snnz, // number of nonzeros in S (stored in CSR, full sparse)
    int *Srowptr, // row pointer S
    int *Srowidx, // row index S 
    int *Scolidx, //col index S 
    double *Sval, // val array S
    double *tval 
    ){
    
    int threadidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadidx < Snnz) {
        int i, j, il, is, jl, js;
        double Lii, s=0.0, sp;

        // handle element S(threadidx) = S(i,j)
        i = Srowidx[ threadidx ];
        j = Scolidx[ threadidx ];


	// we are working on a symmetric matrix S
        // if we notice j>i, we compute S(j,i) instead of S(i,j)
        // maybe later
        if( i>j ){
	//	return;
	    // swap indices - there might be a more efficient way, though
            int t = i;
            i = j;
            j = t;
        }

        // retrieve L(i,i), easy as these are the first element in each row
        Lii = Lval[ Lrowptr[ i ] ];
        // compute L(i,:).* S(j,:)
        // il and is are iterating over the nonzero entries in the respective rows
        il = Lrowptr[ i ]+1;
        is = Srowptr[ j ]+1;
        while( il < Lrowptr[i+1] && is < Srowptr[ j+1 ] ){
            sp = 0.0;
            // jl and js are the col-indices of the respective nonzero entries
            jl = Lcolidx[ il ];
            js = Scolidx[ is ];
	    if(jl==js){
	    	//printf("match in i:%d j:%d jl:%d js:%d\n", i,j,jl,js);
	    }
            sp = (jl == js) ? Lval[ il ] * Sval[ is ] : sp;
            s = (jl == js) ? s+sp : s;
            il = (jl <= js) ? il+1 : il;
            is = (jl >= js) ? is+1 : is;
        }
	// printf("(%d,%d) L(%d %d)= %.2e update %.2e\n", i, j, i, i, Lii, s);
        //s -= sp;  // undo the last operation (it must be the last)    
        s = 1. / Lii * s; // scaling
        
        if (i == j) // diagonal element
            Sval[ threadidx ] = 1. / ( Lii * Lii) - s;
        else  
            Sval[ threadidx ] = - s;



    }
}



void parsinv(
    int n, // matrix size
    int Lnnz, // number of nonzeros in LT stored in CSR, upper triangular  (equivalent to L in CSC)
    int *Lrowptr, // row pointer L
    int *Lcolidx, //col index L
    double *Lval, // val array L
    int Snnz, // number of nonzeros in S (stored in CSR, full sparse)
    int *Srowptr, // row pointer S
    int *Srowidx, // row index S
    int *Scolidx, //col index S
    double *Sval // val array S
    ){
    unsigned int grid_dim = (Snnz + default_block_size - 1) / default_block_size;
    double *tval;
    hipMalloc(&tval, sizeof(double)*Snnz);

    parsinv_kernel<<<dim3(grid_dim), dim3(default_block_size)>>>(n, Lnnz, Lrowptr, Lcolidx, Lval, Snnz, Srowptr, Srowidx, Scolidx, Sval, tval);
    //cudaMemcpy(Sval, tval, sizeof(double)*Snnz, cudaMemcpyDeviceToDevice);
    hipFree(tval);
}
