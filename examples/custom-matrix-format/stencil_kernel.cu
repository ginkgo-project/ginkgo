#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <cstdlib>

#include <ginkgo/ginkgo.hpp>


#define INSTANTIATE_FOR_EACH_VALUE_TYPE(_macro) \
    template _macro(float);                     \
    template _macro(double);


#define STENCIL_KERNEL(_type)                                                 \
    void stencil_kernel(std::size_t size, const _type *coefs, const _type *b, \
                        _type *x);


namespace {


// a parallel CUDA kernel that computes the application of a 3 point stencil
template <typename ValueType>
__global__ void stencil_kernel_impl(std::size_t size, const ValueType *coefs,
                                    const ValueType *b, ValueType *x)
{
    const auto thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= size) {
        return;
    }
    auto result = coefs[1] * b[thread_id];
    if (thread_id > 0) {
        result += coefs[0] * b[thread_id - 1];
    }
    if (thread_id < size - 1) {
        result += coefs[2] * b[thread_id + 1];
    }
    x[thread_id] = result;
}


}  // namespace


template <typename ValueType>
void stencil_kernel(std::size_t size, const ValueType *coefs,
                    const ValueType *b, ValueType *x)
{
    constexpr auto block_size = 512;
    const auto grid_size = (size + block_size - 1) / block_size;
    stencil_kernel_impl<<<grid_size, block_size>>>(size, coefs, b, x);
}

INSTANTIATE_FOR_EACH_VALUE_TYPE(STENCIL_KERNEL);
