/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2022, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "benchmark/utils/timer_impl.hpp"


/**
 * CudaTimer uses cuda executor and cudaEvent to measure the timing.
 */
class CudaTimer : public Timer {
public:
    /**
     * Create a CudaTimer.
     *
     * @param exec  Executor which should be a CudaExecutor
     */
    CudaTimer(std::shared_ptr<const gko::Executor> exec)
        : CudaTimer(std::dynamic_pointer_cast<const gko::CudaExecutor>(exec))
    {}

    /**
     * Create a CudaTimer.
     *
     * @param exec  CudaExecutor associated to the timer
     */
    CudaTimer(std::shared_ptr<const gko::CudaExecutor> exec) : Timer()
    {
        assert(exec != nullptr);
        exec_ = exec;
        auto guard = exec_->get_scoped_device_id_guard();
        GKO_ASSERT_NO_CUDA_ERRORS(hipEventCreate(&start_));
        GKO_ASSERT_NO_CUDA_ERRORS(hipEventCreate(&stop_));
    }

protected:
    void tic_impl() override
    {
        exec_->synchronize();
        auto guard = exec_->get_scoped_device_id_guard();
        // Currently, gko::CudaExecutor always use default stream.
        GKO_ASSERT_NO_CUDA_ERRORS(hipEventRecord(start_));
    }

    double toc_impl() override
    {
        auto guard = exec_->get_scoped_device_id_guard();
        // Currently, gko::CudaExecutor always use default stream.
        GKO_ASSERT_NO_CUDA_ERRORS(hipEventRecord(stop_));
        GKO_ASSERT_NO_CUDA_ERRORS(hipEventSynchronize(stop_));
        float duration_time = 0;
        // hipEventElapsedTime gives the duration_time in milliseconds with a
        // resolution of around 0.5 microseconds
        GKO_ASSERT_NO_CUDA_ERRORS(
            hipEventElapsedTime(&duration_time, start_, stop_));
        constexpr int sec_in_ms = 1e3;
        return static_cast<double>(duration_time) / sec_in_ms;
    }

private:
    std::shared_ptr<const gko::CudaExecutor> exec_;
    hipEvent_t start_;
    hipEvent_t stop_;
};


std::shared_ptr<Timer> get_cuda_timer(
    std::shared_ptr<const gko::CudaExecutor> exec)
{
    return std::make_shared<CudaTimer>(exec);
}
