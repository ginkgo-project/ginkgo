/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2022, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <ginkgo/core/base/executor.hpp>


#include <memory>
#include <type_traits>


#include <gtest/gtest.h>


#include <ginkgo/core/base/exception.hpp>
#include <ginkgo/core/base/exception_helpers.hpp>

#include "common/cuda_hip/base/executor.hpp.inc"
#include "cuda/test/utils.hpp"


namespace {


class ExampleOperation : public gko::Operation {
public:
    explicit ExampleOperation(int& val) : value(val) {}

    void run(std::shared_ptr<const gko::OmpExecutor>) const override
    {
        value = -1;
    }

    void run(std::shared_ptr<const gko::ReferenceExecutor>) const override
    {
        value = -2;
    }

    void run(std::shared_ptr<const gko::HipExecutor>) const override
    {
        value = -3;
    }

    void run(std::shared_ptr<const gko::DpcppExecutor>) const override
    {
        value = -4;
    }

    void run(std::shared_ptr<const gko::CudaExecutor>) const override
    {
        hipGetDevice(&value);
    }

    int& value;
};


class CudaExecutor : public ::testing::Test {
protected:
    CudaExecutor()
        : omp(gko::OmpExecutor::create()),
          cuda(nullptr),
          cuda2(nullptr),
          cuda3(nullptr)
    {}

    void SetUp()
    {
        ASSERT_GT(gko::CudaExecutor::get_num_devices(), 0);
        cuda = gko::CudaExecutor::create(0, omp);
        cuda2 = gko::CudaExecutor::create(
            gko::CudaExecutor::get_num_devices() - 1, omp);
        cuda3 = gko::CudaExecutor::create(0, omp, false,
                                          gko::allocation_mode::unified_global);
    }

    void TearDown()
    {
        if (cuda != nullptr) {
            // ensure that previous calls finished and didn't throw an error
            ASSERT_NO_THROW(cuda->synchronize());
        }
    }

    std::shared_ptr<gko::Executor> omp;
    std::shared_ptr<gko::CudaExecutor> cuda;
    std::shared_ptr<gko::CudaExecutor> cuda2;
    std::shared_ptr<gko::CudaExecutor> cuda3;
};


TEST_F(CudaExecutor, CanInstantiateTwoExecutorsOnOneDevice)
{
    auto cuda = gko::CudaExecutor::create(0, omp);
    auto cuda2 = gko::CudaExecutor::create(0, omp);

    // We want automatic deinitialization to not create any error
}


TEST_F(CudaExecutor, MasterKnowsNumberOfDevices)
{
    int count = 0;
    hipGetDeviceCount(&count);

    auto num_devices = gko::CudaExecutor::get_num_devices();

    ASSERT_EQ(count, num_devices);
}


/* Properly checks if it works only when multiple GPUs exist */
TEST_F(CudaExecutor, PreservesDeviceSettings)
{
    auto previous_device = gko::CudaExecutor::get_num_devices() - 1;
    GKO_ASSERT_NO_CUDA_ERRORS(hipSetDevice(previous_device));
    auto orig = cuda->get_mem_space()->alloc<int>(2);
    int current_device;
    GKO_ASSERT_NO_CUDA_ERRORS(hipGetDevice(&current_device));
    ASSERT_EQ(current_device, previous_device);

    cuda->get_mem_space()->free(orig);
    GKO_ASSERT_NO_CUDA_ERRORS(hipGetDevice(&current_device));
    ASSERT_EQ(current_device, previous_device);
}


TEST_F(CudaExecutor, RunsOnProperDevice)
{
    int value = -1;

    GKO_ASSERT_NO_CUDA_ERRORS(hipSetDevice(0));
    cuda2->run(ExampleOperation(value));

    ASSERT_EQ(value, cuda2->get_device_id());
}


TEST_F(CudaExecutor, Synchronizes)
{
    // Todo design a proper unit test once we support streams
    ASSERT_NO_THROW(cuda->synchronize());
}


TEST_F(CudaExecutor, ExecInfoSetsCorrectProperties)
{
    auto dev_id = cuda->get_device_id();
    auto num_sm = 0;
    auto major = 0;
    auto minor = 0;
    auto max_threads_per_block = 0;
    auto warp_size = 0;
    GKO_ASSERT_NO_CUDA_ERRORS(hipDeviceGetAttribute(
        &num_sm, hipDeviceAttributeMultiprocessorCount, dev_id));
    GKO_ASSERT_NO_CUDA_ERRORS(hipDeviceGetAttribute(
        &major, hipDeviceAttributeComputeCapabilityMajor, dev_id));
    GKO_ASSERT_NO_CUDA_ERRORS(hipDeviceGetAttribute(
        &minor, hipDeviceAttributeComputeCapabilityMinor, dev_id));
    GKO_ASSERT_NO_CUDA_ERRORS(hipDeviceGetAttribute(
        &max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, dev_id));
    GKO_ASSERT_NO_CUDA_ERRORS(
        hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, dev_id));
    auto num_cores = convert_sm_ver_to_cores(major, minor);

    ASSERT_EQ(cuda->get_major_version(), major);
    ASSERT_EQ(cuda->get_minor_version(), minor);
    ASSERT_EQ(cuda->get_num_multiprocessor(), num_sm);
    ASSERT_EQ(cuda->get_warp_size(), warp_size);
    ASSERT_EQ(cuda->get_num_warps(), num_sm * (num_cores / warp_size));
    ASSERT_EQ(cuda->get_num_warps_per_sm(), num_cores / warp_size);
}


}  // namespace
