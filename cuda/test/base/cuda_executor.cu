/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2019, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <ginkgo/core/base/executor.hpp>


#include <memory>
#include <type_traits>


#include <gtest/gtest.h>


#include <ginkgo/core/base/exception.hpp>
#include <ginkgo/core/base/exception_helpers.hpp>


namespace {


class ExampleOperation : public gko::Operation {
public:
    explicit ExampleOperation(int &val) : value(val) {}
    void run(std::shared_ptr<const gko::OmpExecutor>) const override
    {
        value = -1;
    }
    void run(std::shared_ptr<const gko::CudaExecutor>) const override
    {
        hipGetDevice(&value);
    }
    void run(std::shared_ptr<const gko::ReferenceExecutor>) const override
    {
        value = -2;
    }

    int &value;
};


class CudaExecutor : public ::testing::Test {
protected:
    CudaExecutor()
        : omp(gko::OmpExecutor::create()), cuda(nullptr), cuda2(nullptr)
    {}

    void SetUp()
    {
        ASSERT_GT(gko::CudaExecutor::get_num_devices(), 0);
        cuda = gko::CudaExecutor::create(0, omp);
        cuda2 = gko::CudaExecutor::create(
            gko::CudaExecutor::get_num_devices() - 1, omp);
    }

    void TearDown()
    {
        if (cuda != nullptr) {
            // ensure that previous calls finished and didn't throw an error
            ASSERT_NO_THROW(cuda->synchronize());
        }
    }

    std::shared_ptr<gko::Executor> omp;
    std::shared_ptr<gko::CudaExecutor> cuda;
    std::shared_ptr<gko::CudaExecutor> cuda2;
};


TEST_F(CudaExecutor, CanInstantiateTwoExecutorsOnOneDevice)
{
    auto cuda = gko::CudaExecutor::create(0, omp);
    auto cuda2 = gko::CudaExecutor::create(0, omp);

    // We want automatic deinitialization to not create any error
}


TEST_F(CudaExecutor, CanBeCreatedWithAssociatedMemorySpace)
{
    auto mem_space = gko::CudaMemorySpace::create(0);
    auto cuda =
        gko::CudaExecutor::create(0, mem_space, gko::OmpExecutor::create());
    auto cuda2 = gko::CudaExecutor::create(0, omp);

    ASSERT_NE(cuda->get_mem_space(), cuda2->get_mem_space());
    ASSERT_EQ(cuda->get_mem_space(), mem_space);
}


TEST_F(CudaExecutor, MasterKnowsNumberOfDevices)
{
    int count = 0;
    hipGetDeviceCount(&count);

    auto num_devices = gko::CudaExecutor::get_num_devices();

    ASSERT_EQ(count, num_devices);
}


/* Properly checks if it works only when multiple GPUs exist */
TEST_F(CudaExecutor, PreservesDeviceSettings)
{
    auto previous_device = gko::CudaExecutor::get_num_devices() - 1;
    GKO_ASSERT_NO_CUDA_ERRORS(hipSetDevice(previous_device));
    auto orig = cuda->get_mem_space()->alloc<int>(2);
    int current_device;
    GKO_ASSERT_NO_CUDA_ERRORS(hipGetDevice(&current_device));
    ASSERT_EQ(current_device, previous_device);

    cuda->get_mem_space()->free(orig);
    GKO_ASSERT_NO_CUDA_ERRORS(hipGetDevice(&current_device));
    ASSERT_EQ(current_device, previous_device);
}


TEST_F(CudaExecutor, RunsOnProperDevice)
{
    int value = -1;

    GKO_ASSERT_NO_CUDA_ERRORS(hipSetDevice(0));
    cuda2->run(ExampleOperation(value));

    ASSERT_EQ(value, cuda2->get_device_id());
}



TEST_F(CudaExecutor, Synchronizes)
{
    // Todo design a proper unit test once we support streams
    ASSERT_NO_THROW(cuda->synchronize());
}


}  // namespace
