#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2019, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <ginkgo/core/base/executor.hpp>
#include <ginkgo/core/base/memory_space.hpp>


#include <type_traits>


#include <gtest/gtest.h>


#include <ginkgo/core/base/exception.hpp>
#include <ginkgo/core/base/exception_helpers.hpp>


namespace {


class CudaMemorySpace : public ::testing::Test {
protected:
    CudaMemorySpace() : cuda(nullptr), cuda2(nullptr) {}

    void SetUp()
    {
        omp = gko::HostMemorySpace::create();
        cuda = gko::CudaMemorySpace::create(0);
        cuda2 = gko::CudaMemorySpace::create(
            gko::CudaMemorySpace::get_num_devices() - 1);
    }

    void TearDown()
    {
        if (cuda != nullptr) {
            // ensure that previous calls finished and didn't throw an error
            ASSERT_NO_THROW(cuda->synchronize());
        }
    }

    std::shared_ptr<gko::HostMemorySpace> omp;
    std::shared_ptr<gko::CudaMemorySpace> cuda;
    std::shared_ptr<gko::CudaMemorySpace> cuda2;
};


TEST_F(CudaMemorySpace, AllocatesAndFreesMemory)
{
    int *ptr = nullptr;

    ASSERT_NO_THROW(ptr = cuda->alloc<int>(2));
    ASSERT_NO_THROW(cuda->free(ptr));
}


TEST_F(CudaMemorySpace, FailsWhenOverallocating)
{
    const gko::size_type num_elems = 1ll << 50;  // 4PB of integers
    int *ptr = nullptr;

    ASSERT_THROW(
        {
            ptr = cuda->alloc<int>(num_elems);
            cuda->synchronize();
        },
        gko::AllocationError);

    cuda->free(ptr);
}


__global__ void check_data(int *data)
{
    if (data[0] != 3 || data[1] != 8) {
        asm("trap;");
    }
}


TEST_F(CudaMemorySpace, CopiesDataToCuda)
{
    int orig[] = {3, 8};
    auto *copy = cuda->alloc<int>(2);

    cuda->copy_from(omp.get(), 2, orig, copy);

    check_data<<<1, 1>>>(copy);
    ASSERT_NO_THROW(cuda->synchronize());
    cuda->free(copy);
}


__global__ void init_data(int *data)
{
    data[0] = 3;
    data[1] = 8;
}

TEST_F(CudaMemorySpace, CopiesDataFromCuda)
{
    int copy[2];
    auto orig = cuda->alloc<int>(2);
    init_data<<<1, 1>>>(orig);

    omp->copy_from(cuda.get(), 2, orig, copy);

    EXPECT_EQ(3, copy[0]);
    ASSERT_EQ(8, copy[1]);
    cuda->free(orig);
}


TEST_F(CudaMemorySpace, CopiesDataFromCudaToCuda)
{
    int copy[2];
    auto orig = cuda->alloc<int>(2);
    GKO_ASSERT_NO_CUDA_ERRORS(hipSetDevice(0));
    init_data<<<1, 1>>>(orig);

    auto copy_cuda2 = cuda2->alloc<int>(2);
    cuda2->copy_from(cuda.get(), 2, orig, copy_cuda2);

    // Check that the data is really on GPU2 and ensure we did not cheat
    GKO_ASSERT_NO_CUDA_ERRORS(hipSetDevice(cuda2->get_device_id()));
    check_data<<<1, 1>>>(copy_cuda2);
    GKO_ASSERT_NO_CUDA_ERRORS(hipSetDevice(0));

    omp->copy_from(cuda2.get(), 2, copy_cuda2, copy);

    EXPECT_EQ(3, copy[0]);
    ASSERT_EQ(8, copy[1]);
    cuda->free(copy_cuda2);
    cuda->free(orig);
}


}  // namespace
