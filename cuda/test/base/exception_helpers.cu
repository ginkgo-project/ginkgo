// SPDX-FileCopyrightText: 2017-2023 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include <ginkgo/core/base/exception_helpers.hpp>


#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>
#include <hipsparse.h>


#include <gtest/gtest.h>


namespace {


TEST(AssertNoCudaErrors, ThrowsOnError)
{
    ASSERT_THROW(GKO_ASSERT_NO_CUDA_ERRORS(1), gko::CudaError);
}


TEST(AssertNoCudaErrors, DoesNotThrowOnSuccess)
{
    ASSERT_NO_THROW(GKO_ASSERT_NO_CUDA_ERRORS(hipSuccess));
}


TEST(AssertNoCublasErrors, ThrowsOnError)
{
    ASSERT_THROW(GKO_ASSERT_NO_CUBLAS_ERRORS(1), gko::CublasError);
}


TEST(AssertNoCublasErrors, DoesNotThrowOnSuccess)
{
    ASSERT_NO_THROW(GKO_ASSERT_NO_CUBLAS_ERRORS(HIPBLAS_STATUS_SUCCESS));
}


TEST(AssertNoCurandErrors, ThrowsOnError)
{
    ASSERT_THROW(GKO_ASSERT_NO_CURAND_ERRORS(1), gko::CurandError);
}


TEST(AssertNoCurandErrors, DoesNotThrowOnSuccess)
{
    ASSERT_NO_THROW(GKO_ASSERT_NO_CURAND_ERRORS(HIPRAND_STATUS_SUCCESS));
}


TEST(AssertNoCusparseErrors, ThrowsOnError)
{
    ASSERT_THROW(GKO_ASSERT_NO_CUSPARSE_ERRORS(1), gko::CusparseError);
}


TEST(AssertNoCusparseErrors, DoesNotThrowOnSuccess)
{
    ASSERT_NO_THROW(GKO_ASSERT_NO_CUSPARSE_ERRORS(HIPSPARSE_STATUS_SUCCESS));
}


TEST(AssertNoCufftErrors, ThrowsOnError)
{
    ASSERT_THROW(GKO_ASSERT_NO_CUFFT_ERRORS(1), gko::CufftError);
}


TEST(AssertNoCufftErrors, DoesNotThrowOnSuccess)
{
    ASSERT_NO_THROW(GKO_ASSERT_NO_CUFFT_ERRORS(HIPFFT_SUCCESS));
}


}  // namespace
