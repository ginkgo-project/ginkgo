/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/preconditioner/jacobi_kernels.hpp"


#include <ginkgo/config.hpp>
#include <ginkgo/core/base/exception_helpers.hpp>


#include "core/base/extended_float.hpp"
#include "core/components/fill_array.hpp"
#include "core/preconditioner/jacobi_utils.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/diagonal_block_manipulation.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/components/uninitialized_array.hpp"
#include "cuda/components/warp_blas.cuh"
#include "cuda/preconditioner/jacobi_common.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The Jacobi preconditioner namespace.
 * @ref Jacobi
 * @ingroup jacobi
 */
namespace jacobi {


#include "common/preconditioner/jacobi_generate_kernel.hpp.inc"


namespace {


template <int warps_per_block, int max_block_size, typename ValueType,
          typename IndexType>
void generate(syn::value_list<int, max_block_size>,
              const matrix::Csr<ValueType, IndexType> *mtx,
              remove_complex<ValueType> accuracy, ValueType *block_data,
              const preconditioner::block_interleaved_storage_scheme<IndexType>
                  &storage_scheme,
              remove_complex<ValueType> *conditioning,
              precision_reduction *block_precisions,
              const IndexType *block_ptrs, size_type num_blocks)
{
    constexpr int subwarp_size = get_larger_power(max_block_size);
    constexpr int blocks_per_warp = config::warp_size / subwarp_size;
    const dim3 grid_size(ceildiv(num_blocks, warps_per_block * blocks_per_warp),
                         1, 1);
    const dim3 block_size(subwarp_size, blocks_per_warp, warps_per_block);

    if (block_precisions) {
        kernel::adaptive_generate<max_block_size, subwarp_size, warps_per_block>
            <<<grid_size, block_size, 0, 0>>>(
                mtx->get_size()[0], mtx->get_const_row_ptrs(),
                mtx->get_const_col_idxs(),
                as_cuda_type(mtx->get_const_values()), as_cuda_type(accuracy),
                as_cuda_type(block_data), storage_scheme,
                as_cuda_type(conditioning), block_precisions, block_ptrs,
                num_blocks);
    } else {
        kernel::generate<max_block_size, subwarp_size, warps_per_block>
            <<<grid_size, block_size, 0, 0>>>(
                mtx->get_size()[0], mtx->get_const_row_ptrs(),
                mtx->get_const_col_idxs(),
                as_cuda_type(mtx->get_const_values()), as_cuda_type(block_data),
                storage_scheme, block_ptrs, num_blocks);
    }
}

GKO_ENABLE_IMPLEMENTATION_SELECTION(select_generate, generate);


}  // namespace


template <typename ValueType, typename IndexType>
void generate(std::shared_ptr<const CudaExecutor> exec,
              const matrix::Csr<ValueType, IndexType> *system_matrix,
              size_type num_blocks, uint32 max_block_size,
              remove_complex<ValueType> accuracy,
              const preconditioner::block_interleaved_storage_scheme<IndexType>
                  &storage_scheme,
              Array<remove_complex<ValueType>> &conditioning,
              Array<precision_reduction> &block_precisions,
              const Array<IndexType> &block_pointers, Array<ValueType> &blocks)
{
    components::fill_array(exec, blocks.get_data(), blocks.get_num_elems(),
                           zero<ValueType>());
    select_generate(
        compiled_kernels(),
        [&](int compiled_block_size) {
            return max_block_size <= compiled_block_size;
        },
        syn::value_list<int, config::min_warps_per_block>(), syn::type_list<>(),
        system_matrix, accuracy, blocks.get_data(), storage_scheme,
        conditioning.get_data(), block_precisions.get_data(),
        block_pointers.get_const_data(), num_blocks);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_JACOBI_GENERATE_KERNEL);


}  // namespace jacobi
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
