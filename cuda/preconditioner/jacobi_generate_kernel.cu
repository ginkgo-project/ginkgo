// SPDX-FileCopyrightText: 2017-2023 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/preconditioner/jacobi_kernels.hpp"


#include <ginkgo/config.hpp>
#include <ginkgo/core/base/exception_helpers.hpp>


#include "core/components/fill_array_kernels.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/preconditioner/jacobi_common.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The Jacobi preconditioner namespace.
 * @ref Jacobi
 * @ingroup jacobi
 */
namespace jacobi {


template <int warps_per_block, int max_block_size, typename ValueType,
          typename IndexType>
void generate(syn::value_list<int, max_block_size>,
              std::shared_ptr<const DefaultExecutor> exec,
              const matrix::Csr<ValueType, IndexType>* mtx,
              remove_complex<ValueType> accuracy, ValueType* block_data,
              const preconditioner::block_interleaved_storage_scheme<IndexType>&
                  storage_scheme,
              remove_complex<ValueType>* conditioning,
              precision_reduction* block_precisions,
              const IndexType* block_ptrs, size_type num_blocks);

GKO_ENABLE_IMPLEMENTATION_SELECTION(select_generate, generate);


template <typename ValueType, typename IndexType>
void generate(std::shared_ptr<const CudaExecutor> exec,
              const matrix::Csr<ValueType, IndexType>* system_matrix,
              size_type num_blocks, uint32 max_block_size,
              remove_complex<ValueType> accuracy,
              const preconditioner::block_interleaved_storage_scheme<IndexType>&
                  storage_scheme,
              array<remove_complex<ValueType>>& conditioning,
              array<precision_reduction>& block_precisions,
              const array<IndexType>& block_pointers, array<ValueType>& blocks)
{
    components::fill_array(exec, blocks.get_data(), blocks.get_num_elems(),
                           zero<ValueType>());
    select_generate(
        compiled_kernels(),
        [&](int compiled_block_size) {
            return max_block_size <= compiled_block_size;
        },
        syn::value_list<int, config::min_warps_per_block>(), syn::type_list<>(),
        exec, system_matrix, accuracy, blocks.get_data(), storage_scheme,
        conditioning.get_data(), block_precisions.get_data(),
        block_pointers.get_const_data(), num_blocks);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_JACOBI_GENERATE_KERNEL);


}  // namespace jacobi
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
