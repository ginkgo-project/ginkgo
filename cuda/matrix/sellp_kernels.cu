#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2018

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/matrix/sellp_kernels.hpp"


#include "core/base/exception_helpers.hpp"
#include "core/base/math.hpp"
#include "core/base/types.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
namespace sellp {


namespace {


template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(matrix::default_slice_size) void spmv_kernel(
    size_type num_rows, size_type num_right_hand_sides, size_type b_stride,
    size_type c_stride, const size_type *__restrict__ slice_lengths,
    const size_type *__restrict__ slice_sets, const ValueType *__restrict__ a,
    const IndexType *__restrict__ col, const ValueType *__restrict__ b,
    ValueType *__restrict__ c)
{
    const auto slice_id = blockIdx.x;
    const auto slice_size = blockDim.x;
    const auto row_in_slice = threadIdx.x;
    const auto global_row =
        static_cast<size_type>(slice_size) * slice_id + row_in_slice;
    const auto column_id = blockIdx.y;
    ValueType val = 0;
    IndexType ind = 0;
    if (global_row < num_rows && column_id < num_right_hand_sides) {
        for (size_type i = 0; i < slice_lengths[slice_id]; i++) {
            ind = row_in_slice + (slice_sets[slice_id] + i) * slice_size;
            val += a[ind] * b[col[ind] * b_stride + column_id];
        }
        c[global_row * c_stride + column_id] = val;
    }
}


}  // namespace


template <typename ValueType, typename IndexType>
void spmv(std::shared_ptr<const CudaExecutor> exec,
          const matrix::Sellp<ValueType, IndexType> *a,
          const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *c)
{
    const dim3 blockSize(matrix::default_slice_size);
    const dim3 gridSize(
        ceildiv(a->get_size().num_rows, matrix::default_slice_size),
        b->get_size().num_cols);

    spmv_kernel<<<gridSize, blockSize>>>(
        a->get_size().num_rows, b->get_size().num_cols, b->get_stride(),
        c->get_stride(), a->get_const_slice_lengths(),
        a->get_const_slice_sets(), as_cuda_type(a->get_const_values()),
        a->get_const_col_idxs(), as_cuda_type(b->get_const_values()),
        as_cuda_type(c->get_values()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_SELLP_SPMV_KERNEL);


namespace {


template <typename ValueType, typename IndexType>
__global__
    __launch_bounds__(matrix::default_slice_size) void advanced_spmv_kernel(
        size_type num_rows, size_type num_right_hand_sides, size_type b_stride,
        size_type c_stride, const size_type *__restrict__ slice_lengths,
        const size_type *__restrict__ slice_sets,
        const ValueType *__restrict__ alpha, const ValueType *__restrict__ a,
        const IndexType *__restrict__ col, const ValueType *__restrict__ b,
        const ValueType *__restrict__ beta, ValueType *__restrict__ c)
{
    const auto slice_id = blockIdx.x;
    const auto slice_size = blockDim.x;
    const auto row_in_slice = threadIdx.x;
    const auto global_row =
        static_cast<size_type>(slice_size) * slice_id + row_in_slice;
    const auto column_id = blockIdx.y;
    ValueType val = 0;
    IndexType ind = 0;
    if (global_row < num_rows && column_id < num_right_hand_sides) {
        for (size_type i = 0; i < slice_lengths[slice_id]; i++) {
            ind = row_in_slice + (slice_sets[slice_id] + i) * slice_size;
            val += alpha[0] * a[ind] * b[col[ind] * b_stride + column_id];
        }
        c[global_row * c_stride + column_id] =
            beta[0] * c[global_row * c_stride + column_id] + val;
    }
}


}  // namespace


template <typename ValueType, typename IndexType>
void advanced_spmv(std::shared_ptr<const CudaExecutor> exec,
                   const matrix::Dense<ValueType> *alpha,
                   const matrix::Sellp<ValueType, IndexType> *a,
                   const matrix::Dense<ValueType> *b,
                   const matrix::Dense<ValueType> *beta,
                   matrix::Dense<ValueType> *c)
{
    const dim3 blockSize(matrix::default_slice_size);
    const dim3 gridSize(
        ceildiv(a->get_size().num_rows, matrix::default_slice_size),
        b->get_size().num_cols);

    advanced_spmv_kernel<<<gridSize, blockSize>>>(
        a->get_size().num_rows, b->get_size().num_cols, b->get_stride(),
        c->get_stride(), a->get_const_slice_lengths(),
        a->get_const_slice_sets(), as_cuda_type(alpha->get_const_values()),
        as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
        as_cuda_type(b->get_const_values()),
        as_cuda_type(beta->get_const_values()), as_cuda_type(c->get_values()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SELLP_ADVANCED_SPMV_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_dense(
    std::shared_ptr<const CudaExecutor> exec, matrix::Dense<ValueType> *result,
    const matrix::Sellp<ValueType, IndexType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SELLP_CONVERT_TO_DENSE_KERNEL);


}  // namespace sellp
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
