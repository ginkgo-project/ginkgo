#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2018

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/matrix/dense_kernels.hpp"


#include "core/base/math.hpp"
#include "cuda/base/cublas_bindings.hpp"
#include "cuda/components/reduction.cuh"
#include "cuda/components/uninitialized_array.hpp"


namespace gko {
namespace kernels {
namespace cuda {
namespace dense {


constexpr auto default_block_size = 512;


template <typename ValueType>
void simple_apply(std::shared_ptr<const CudaExecutor> exec,
                  const matrix::Dense<ValueType> *a,
                  const matrix::Dense<ValueType> *b,
                  matrix::Dense<ValueType> *c)
{
    if (cublas::is_supported<ValueType>::value) {
        auto handle = cublas::init();
        ASSERT_NO_CUBLAS_ERRORS(
            hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));
        auto alpha = one<ValueType>();
        auto beta = zero<ValueType>();
        cublas::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, c->get_size()[1],
                     c->get_size()[0], a->get_size()[1], &alpha,
                     b->get_const_values(), b->get_stride(),
                     a->get_const_values(), a->get_stride(), &beta,
                     c->get_values(), c->get_stride());
        cublas::destroy(handle);
    } else {
        NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_DENSE_SIMPLE_APPLY_KERNEL);


template <typename ValueType>
void apply(std::shared_ptr<const CudaExecutor> exec,
           const matrix::Dense<ValueType> *alpha,
           const matrix::Dense<ValueType> *a, const matrix::Dense<ValueType> *b,
           const matrix::Dense<ValueType> *beta, matrix::Dense<ValueType> *c)
{
    if (cublas::is_supported<ValueType>::value) {
        auto handle = cublas::init();
        cublas::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, c->get_size()[1],
                     c->get_size()[0], a->get_size()[1],
                     alpha->get_const_values(), b->get_const_values(),
                     b->get_stride(), a->get_const_values(), a->get_stride(),
                     beta->get_const_values(), c->get_values(),
                     c->get_stride());
        cublas::destroy(handle);
    } else {
        NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_DENSE_APPLY_KERNEL);


namespace kernel {


template <size_type block_size, typename ValueType>
__global__ __launch_bounds__(block_size) void scale(
    size_type num_rows, size_type num_cols, size_type num_alpha_cols,
    const ValueType *__restrict__ alpha, ValueType *__restrict__ x,
    size_type stride_x)
{
    constexpr auto warps_per_block = block_size / cuda_config::warp_size;
    const auto global_id =
        thread::get_thread_id<cuda_config::warp_size, warps_per_block>();
    const auto row_id = global_id / num_cols;
    const auto col_id = global_id % num_cols;
    const auto alpha_id = num_alpha_cols == 1 ? 0 : col_id;
    if (row_id < num_rows) {
        x[row_id * stride_x + col_id] =
            alpha[alpha_id] == zero<ValueType>()
                ? zero<ValueType>()
                : x[row_id * stride_x + col_id] * alpha[alpha_id];
    }
}


}  // namespace kernel


template <typename ValueType>
void scale(std::shared_ptr<const CudaExecutor> exec,
           const matrix::Dense<ValueType> *alpha, matrix::Dense<ValueType> *x)
{
    if (cublas::is_supported<ValueType>::value && x->get_size()[1] == 1) {
        auto handle = cublas::init();
        cublas::scal(handle, x->get_size()[0], alpha->get_const_values(),
                     x->get_values(), x->get_stride());
        cublas::destroy(handle);
    } else {
        // TODO: tune this parameter
        constexpr auto block_size = default_block_size;
        const dim3 grid_dim =
            ceildiv(x->get_size()[0] * x->get_size()[1], block_size);
        const dim3 block_dim{cuda_config::warp_size, 1,
                             block_size / cuda_config::warp_size};
        kernel::scale<block_size><<<grid_dim, block_dim>>>(
            x->get_size()[0], x->get_size()[1], alpha->get_size()[1],
            as_cuda_type(alpha->get_const_values()),
            as_cuda_type(x->get_values()), x->get_stride());
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_DENSE_SCALE_KERNEL);


namespace kernel {


template <size_type block_size, typename ValueType>
__global__ __launch_bounds__(block_size) void add_scaled(
    size_type num_rows, size_type num_cols, size_type num_alpha_cols,
    const ValueType *__restrict__ alpha, const ValueType *__restrict__ x,
    size_type stride_x, ValueType *__restrict__ y, size_type stride_y)
{
    constexpr auto warps_per_block = block_size / cuda_config::warp_size;
    const auto global_id =
        thread::get_thread_id<cuda_config::warp_size, warps_per_block>();
    const auto row_id = global_id / num_cols;
    const auto col_id = global_id % num_cols;
    const auto alpha_id = num_alpha_cols == 1 ? 0 : col_id;
    if (row_id < num_rows && alpha[alpha_id] != zero<ValueType>()) {
        y[row_id * stride_y + col_id] +=
            x[row_id * stride_x + col_id] * alpha[alpha_id];
    }
}


}  // namespace kernel


template <typename ValueType>
void add_scaled(std::shared_ptr<const CudaExecutor> exec,
                const matrix::Dense<ValueType> *alpha,
                const matrix::Dense<ValueType> *x, matrix::Dense<ValueType> *y)
{
    if (cublas::is_supported<ValueType>::value && x->get_size()[1] == 1) {
        auto handle = cublas::init();
        cublas::axpy(handle, x->get_size()[0], alpha->get_const_values(),
                     x->get_const_values(), x->get_stride(), y->get_values(),
                     y->get_stride());
        cublas::destroy(handle);
    } else {
        // TODO: tune this parameter
        constexpr auto block_size = default_block_size;
        const dim3 grid_dim =
            ceildiv(x->get_size()[0] * x->get_size()[1], block_size);
        const dim3 block_dim{cuda_config::warp_size, 1,
                             block_size / cuda_config::warp_size};
        kernel::add_scaled<block_size><<<grid_dim, block_dim>>>(
            x->get_size()[0], x->get_size()[1], alpha->get_size()[1],
            as_cuda_type(alpha->get_const_values()),
            as_cuda_type(x->get_const_values()), x->get_stride(),
            as_cuda_type(y->get_values()), y->get_stride());
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_DENSE_ADD_SCALED_KERNEL);


namespace kernel {


template <size_type block_size, typename ValueType>
__global__ __launch_bounds__(block_size) void compute_partial_dot(
    size_type num_rows, const ValueType *__restrict__ x, size_type stride_x,
    const ValueType *__restrict__ y, size_type stride_y,
    ValueType *__restrict__ work)
{
    constexpr auto warps_per_block = block_size / cuda_config::warp_size;

    const auto num_blocks = gridDim.x;
    const auto local_id = thread::get_local_thread_id<cuda_config::warp_size>();
    const auto global_id =
        thread::get_thread_id<cuda_config::warp_size, warps_per_block>();

    auto tmp = zero<ValueType>();
    for (auto i = global_id; i < num_rows; i += block_size * num_blocks) {
        tmp += x[i * stride_x] * y[i * stride_y];
    }
    __shared__ UninitializedArray<ValueType, block_size> tmp_work;
    tmp_work[local_id] = tmp;

    block::reduce<block_size, cuda_config::warp_size>(
        static_cast<ValueType *>(tmp_work),
        [](const ValueType &x, const ValueType &y) { return x + y; });

    if (local_id == 0) {
        work[thread::get_block_id()] = tmp_work[0];
    }
}


template <size_type block_size, typename ValueType>
__global__ __launch_bounds__(block_size) void finalize_dot_computation(
    size_type size, const ValueType *work, ValueType *result)
{
    const auto local_id = thread::get_local_thread_id<cuda_config::warp_size>();

    ValueType tmp = zero<ValueType>();
    for (auto i = local_id; i < size; i += block_size) {
        tmp += work[i];
    }
    __shared__ UninitializedArray<ValueType, block_size> tmp_work;
    tmp_work[local_id] = tmp;

    block::reduce<block_size, cuda_config::warp_size>(
        static_cast<ValueType *>(tmp_work),
        [](const ValueType &x, const ValueType &y) { return x + y; });

    if (local_id == 0) {
        *result = tmp_work[0];
    }
}


}  // namespace kernel


template <typename ValueType>
void compute_dot(std::shared_ptr<const CudaExecutor> exec,
                 const matrix::Dense<ValueType> *x,
                 const matrix::Dense<ValueType> *y,
                 matrix::Dense<ValueType> *result)
{
    if (cublas::is_supported<ValueType>::value) {
        auto handle = cublas::init();
        // TODO: write a custom kernel which does this more efficiently
        for (size_type col = 0; col < x->get_size()[1]; ++col) {
            cublas::dot(handle, x->get_size()[0], x->get_const_values() + col,
                        x->get_stride(), y->get_const_values() + col,
                        y->get_stride(), result->get_values() + col);
        }
        cublas::destroy(handle);
    } else {
        // TODO: these are tuning parameters obtained experimentally, once
        // we decide how to handle this uniformly, they should be modified
        // appropriately
        constexpr auto work_per_thread = 32;
        constexpr auto block_size = 1024;

        constexpr auto work_per_block = work_per_thread * block_size;
        const dim3 grid_dim = ceildiv(x->get_size()[0], work_per_block);
        const dim3 block_dim{cuda_config::warp_size, 1,
                             block_size / cuda_config::warp_size};
        Array<ValueType> work(exec, grid_dim.x);
        // TODO: write a kernel which does this more efficiently
        for (size_type col = 0; col < x->get_size()[1]; ++col) {
            kernel::compute_partial_dot<block_size><<<grid_dim, block_dim>>>(
                x->get_size()[0], as_cuda_type(x->get_const_values() + col),
                x->get_stride(), as_cuda_type(y->get_const_values() + col),
                y->get_stride(), as_cuda_type(work.get_data()));
            kernel::finalize_dot_computation<block_size><<<1, block_dim>>>(
                grid_dim.x, as_cuda_type(work.get_const_data()),
                as_cuda_type(result->get_values() + col));
        }
    }
}


GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_DENSE_COMPUTE_DOT_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_coo(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Coo<ValueType, IndexType> *result,
                    const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_CONVERT_TO_COO_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_csr(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Csr<ValueType, IndexType> *result,
                    const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_CONVERT_TO_CSR_KERNEL);


template <typename ValueType, typename IndexType>
void move_to_csr(std::shared_ptr<const CudaExecutor> exec,
                 matrix::Csr<ValueType, IndexType> *result,
                 const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_MOVE_TO_CSR_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_ell(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Ell<ValueType, IndexType> *result,
                    const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_CONVERT_TO_ELL_KERNEL);


template <typename ValueType, typename IndexType>
void move_to_ell(std::shared_ptr<const CudaExecutor> exec,
                 matrix::Ell<ValueType, IndexType> *result,
                 const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_MOVE_TO_ELL_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_hybrid(std::shared_ptr<const CudaExecutor> exec,
                       matrix::Hybrid<ValueType, IndexType> *result,
                       const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_CONVERT_TO_HYBRID_KERNEL);


template <typename ValueType, typename IndexType>
void move_to_hybrid(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Hybrid<ValueType, IndexType> *result,
                    const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_MOVE_TO_HYBRID_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_sellp(std::shared_ptr<const CudaExecutor> exec,
                      matrix::Sellp<ValueType, IndexType> *result,
                      const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_CONVERT_TO_SELLP_KERNEL);


template <typename ValueType, typename IndexType>
void move_to_sellp(std::shared_ptr<const CudaExecutor> exec,
                   matrix::Sellp<ValueType, IndexType> *result,
                   const matrix::Dense<ValueType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_DENSE_MOVE_TO_SELLP_KERNEL);


template <typename ValueType>
void count_nonzeros(std::shared_ptr<const CudaExecutor> exec,
                    const matrix::Dense<ValueType> *source,
                    size_type *result) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_DENSE_COUNT_NONZEROS_KERNEL);


template <typename ValueType>
void calculate_max_nnz_per_row(std::shared_ptr<const CudaExecutor> exec,
                               const matrix::Dense<ValueType> *source,
                               size_type *result) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_DENSE_CALCULATE_MAX_NNZ_PER_ROW_KERNEL);


template <typename ValueType>
void calculate_nonzeros_per_row(std::shared_ptr<const CudaExecutor> exec,
                                const matrix::Dense<ValueType> *source,
                                Array<size_type> *result) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_DENSE_CALCULATE_NONZEROS_PER_ROW_KERNEL);


template <typename ValueType>
void calculate_total_cols(std::shared_ptr<const CudaExecutor> exec,
                          const matrix::Dense<ValueType> *source,
                          size_type *result,
                          size_type stride_factor) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_DENSE_CALCULATE_TOTAL_COLS_KERNEL);


template <typename ValueType>
void transpose(std::shared_ptr<const CudaExecutor> exec,
               matrix::Dense<ValueType> *trans,
               const matrix::Dense<ValueType> *orig)
{
    if (cublas::is_supported<ValueType>::value) {
        auto handle = cublas::init();
        ASSERT_NO_CUBLAS_ERRORS(
            hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));

        auto alpha = one<ValueType>();
        auto beta = zero<ValueType>();
        cublas::geam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, orig->get_size()[0],
                     orig->get_size()[1], &alpha, orig->get_const_values(),
                     orig->get_stride(), &beta,
                     static_cast<ValueType *>(nullptr), trans->get_size()[1],
                     trans->get_values(), trans->get_stride());

        cublas::destroy(handle);
    } else {
        NOT_IMPLEMENTED;
    }
};

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_TRANSPOSE_KERNEL);


template <typename ValueType>
void conj_transpose(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Dense<ValueType> *trans,
                    const matrix::Dense<ValueType> *orig)

{
    if (cublas::is_supported<ValueType>::value) {
        auto handle = cublas::init();
        ASSERT_NO_CUBLAS_ERRORS(
            hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));

        auto alpha = one<ValueType>();
        auto beta = zero<ValueType>();
        cublas::geam(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, orig->get_size()[0],
                     orig->get_size()[1], &alpha, orig->get_const_values(),
                     orig->get_stride(), &beta,
                     static_cast<ValueType *>(nullptr), trans->get_size()[1],
                     trans->get_values(), trans->get_stride());

        cublas::destroy(handle);
    } else {
        NOT_IMPLEMENTED;
    }
};

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CONJ_TRANSPOSE_KERNEL);


}  // namespace dense
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
