#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2018

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/matrix/ell_kernels.hpp"


#include "core/base/exception_helpers.hpp"
#include "core/base/math.hpp"
#include "core/base/types.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
namespace ell {


constexpr int default_block_size = 512;


namespace {


template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(default_block_size) void spmv_kernel(
    const size_type num_rows, const ValueType *__restrict__ val,
    const IndexType *__restrict__ col, const size_type stride,
    const size_type num_stored_elements_per_row,
    const ValueType *__restrict__ b, const size_type b_stride,
    ValueType *__restrict__ c, const size_type c_stride)
{
    const auto tidx =
        static_cast<IndexType>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto column_id = blockIdx.y;
    ValueType temp = zero<ValueType>();
    IndexType ind = tidx;
    const IndexType finish = ind + num_stored_elements_per_row * stride;
    if (tidx < num_rows) {
        for (; ind < finish; ind += stride) {
            temp += val[ind] * b[col[ind] * b_stride + column_id];
        }
        c[tidx * c_stride + column_id] = temp;
    }
}


}  // namespace


template <typename ValueType, typename IndexType>
void spmv(std::shared_ptr<const CudaExecutor> exec,
          const matrix::Ell<ValueType, IndexType> *a,
          const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *c)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(ceildiv(a->get_size().num_rows, block_size.x),
                         b->get_size().num_cols, 1);

    spmv_kernel<<<grid_size, block_size, 0, 0>>>(
        a->get_size().num_rows, as_cuda_type(a->get_const_values()),
        a->get_const_col_idxs(), a->get_stride(),
        a->get_num_stored_elements_per_row(),
        as_cuda_type(b->get_const_values()), b->get_stride(),
        as_cuda_type(c->get_values()), c->get_stride());
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_ELL_SPMV_KERNEL);


namespace {


template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(default_block_size) void advanced_spmv_kernel(
    const size_type num_rows, const ValueType *__restrict__ alpha,
    const ValueType *__restrict__ val, const IndexType *__restrict__ col,
    const size_type stride, const size_type num_stored_elements_per_row,
    const ValueType *__restrict__ b, const size_type b_stride,
    const ValueType *__restrict__ beta, ValueType *__restrict__ c,
    const size_type c_stride)
{
    const auto tidx =
        static_cast<IndexType>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto column_id = blockIdx.y;
    ValueType temp = zero<ValueType>();
    IndexType ind = tidx;
    const IndexType finish = ind + num_stored_elements_per_row * stride;
    if (tidx < num_rows) {
        for (; ind < finish; ind += stride) {
            temp += alpha[0] * val[ind] * b[col[ind] * b_stride + column_id];
        }
        c[tidx * c_stride + column_id] =
            beta[0] * c[tidx * c_stride + column_id] + temp;
    }
}


}  // namespace


template <typename ValueType, typename IndexType>
void advanced_spmv(std::shared_ptr<const CudaExecutor> exec,
                   const matrix::Dense<ValueType> *alpha,
                   const matrix::Ell<ValueType, IndexType> *a,
                   const matrix::Dense<ValueType> *b,
                   const matrix::Dense<ValueType> *beta,
                   matrix::Dense<ValueType> *c)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(ceildiv(a->get_size().num_rows, block_size.x),
                         b->get_size().num_cols, 1);

    advanced_spmv_kernel<<<grid_size, block_size, 0, 0>>>(
        a->get_size().num_rows, as_cuda_type(alpha->get_const_values()),
        as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
        a->get_stride(), a->get_num_stored_elements_per_row(),
        as_cuda_type(b->get_const_values()), b->get_stride(),
        as_cuda_type(beta->get_const_values()), as_cuda_type(c->get_values()),
        c->get_stride());
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_ELL_ADVANCED_SPMV_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_dense(
    std::shared_ptr<const CudaExecutor> exec, matrix::Dense<ValueType> *result,
    const matrix::Ell<ValueType, IndexType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_ELL_CONVERT_TO_DENSE_KERNEL);


}  // namespace ell
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
