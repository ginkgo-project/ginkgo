// SPDX-FileCopyrightText: 2017-2023 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/matrix/diagonal_kernels.hpp"


#include <ginkgo/core/matrix/csr.hpp>
#include <ginkgo/core/matrix/dense.hpp>


#include "cuda/base/config.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/thread_ids.cuh"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The Diagonal matrix format namespace.
 *
 * @ingroup diagonal
 */
namespace diagonal {


constexpr int default_block_size = 512;


#include "common/cuda_hip/matrix/diagonal_kernels.hpp.inc"


}  // namespace diagonal
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
