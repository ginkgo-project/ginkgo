#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2022, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/matrix/sparsity_csr_kernels.hpp"


#include <ginkgo/core/base/exception_helpers.hpp>


#include "accessor/cuda_helper.hpp"
#include "accessor/reduced_row_major.hpp"
#include "core/base/mixed_precision_types.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/components/uninitialized_array.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The Compressed sparse row matrix format namespace.
 *
 * @ingroup sparsity
 */
namespace sparsity_csr {


constexpr int classical_oversubscription = 32;
constexpr int spmv_block_size = 128;
constexpr int warps_in_block = 4;


using classical_kernels = syn::value_list<int, 2>;


#include "common/cuda_hip/matrix/sparsity_csr_kernels.hpp.inc"


namespace host_kernel {


template <int subwarp_size, typename MatrixValueType, typename InputValueType,
          typename OutputValueType, typename IndexType>
void classical_spmv(syn::value_list<int, subwarp_size>,
                    std::shared_ptr<const CudaExecutor> exec,
                    const matrix::SparsityCsr<MatrixValueType, IndexType>* a,
                    const matrix::Dense<InputValueType>* b,
                    matrix::Dense<OutputValueType>* c,
                    const matrix::Dense<MatrixValueType>* alpha = nullptr,
                    const matrix::Dense<OutputValueType>* beta = nullptr)
{
    using arithmetic_type =
        highest_precision<InputValueType, OutputValueType, MatrixValueType>;
    using input_accessor =
        gko::acc::reduced_row_major<2, arithmetic_type, const InputValueType>;
    using output_accessor =
        gko::acc::reduced_row_major<2, arithmetic_type, OutputValueType>;

    const auto nwarps = exec->get_num_warps_per_sm() *
                        exec->get_num_multiprocessor() *
                        classical_oversubscription;
    const auto gridx =
        std::min(ceildiv(a->get_size()[0], spmv_block_size / subwarp_size),
                 int64(nwarps / warps_in_block));
    const dim3 grid(gridx, b->get_size()[1]);
    const auto block = spmv_block_size;

    const auto b_vals = gko::acc::range<input_accessor>(
        std::array<acc::size_type, 2>{
            {static_cast<acc::size_type>(b->get_size()[0]),
             static_cast<acc::size_type>(b->get_size()[1])}},
        b->get_const_values(),
        std::array<acc::size_type, 1>{
            {static_cast<acc::size_type>(b->get_stride())}});
    auto c_vals = gko::acc::range<output_accessor>(
        std::array<acc::size_type, 2>{
            {static_cast<acc::size_type>(c->get_size()[0]),
             static_cast<acc::size_type>(c->get_size()[1])}},
        c->get_values(),
        std::array<acc::size_type, 1>{
            {static_cast<acc::size_type>(c->get_stride())}});
    if (c->get_size()[0] == 0 || c->get_size()[1] == 0) {
        // empty output: nothing to do
        return;
    }
    if (alpha == nullptr && beta == nullptr) {
        kernel::abstract_classical_spmv<subwarp_size><<<grid, block, 0, 0>>>(
            a->get_size()[0], as_cuda_type(a->get_const_value()),
            a->get_const_col_idxs(), as_cuda_type(a->get_const_row_ptrs()),
            acc::as_cuda_range(b_vals), acc::as_cuda_range(c_vals));
    } else if (alpha != nullptr && beta != nullptr) {
        kernel::abstract_classical_spmv<subwarp_size><<<grid, block, 0, 0>>>(
            a->get_size()[0], as_cuda_type(alpha->get_const_values()),
            as_cuda_type(a->get_const_value()), a->get_const_col_idxs(),
            as_cuda_type(a->get_const_row_ptrs()), acc::as_cuda_range(b_vals),
            as_cuda_type(beta->get_const_values()), acc::as_cuda_range(c_vals));
    } else {
        GKO_KERNEL_NOT_FOUND;
    }
}

GKO_ENABLE_IMPLEMENTATION_SELECTION(select_classical_spmv, classical_spmv);


}  // namespace host_kernel

template <typename MatrixValueType, typename InputValueType,
          typename OutputValueType, typename IndexType>
void spmv(std::shared_ptr<const CudaExecutor> exec,
          const matrix::SparsityCsr<MatrixValueType, IndexType>* a,
          const matrix::Dense<InputValueType>* b,
          matrix::Dense<OutputValueType>* c)
{
    host_kernel::select_classical_spmv(
        classical_kernels(), [](int compiled_info) { return true; },
        syn::value_list<int>(), syn::type_list<>(), exec, a, b, c);
}

GKO_INSTANTIATE_FOR_EACH_MIXED_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_SPMV_KERNEL);


template <typename MatrixValueType, typename InputValueType,
          typename OutputValueType, typename IndexType>
void advanced_spmv(std::shared_ptr<const CudaExecutor> exec,
                   const matrix::Dense<MatrixValueType>* alpha,
                   const matrix::SparsityCsr<MatrixValueType, IndexType>* a,
                   const matrix::Dense<InputValueType>* b,
                   const matrix::Dense<OutputValueType>* beta,
                   matrix::Dense<OutputValueType>* c)
{
    host_kernel::select_classical_spmv(
        classical_kernels(), [](int compiled_info) { return true; },
        syn::value_list<int>(), syn::type_list<>(), exec, a, b, c, alpha, beta);
}

GKO_INSTANTIATE_FOR_EACH_MIXED_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_ADVANCED_SPMV_KERNEL);


template <typename ValueType, typename IndexType>
void fill_in_dense(std::shared_ptr<const DefaultExecutor> exec,
                   const matrix::SparsityCsr<ValueType, IndexType>* input,
                   matrix::Dense<ValueType>* output) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_FILL_IN_DENSE_KERNEL);


template <typename ValueType, typename IndexType>
void count_num_diagonal_elements(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::SparsityCsr<ValueType, IndexType>* matrix,
    size_type* num_diagonal_elements) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_COUNT_NUM_DIAGONAL_ELEMENTS_KERNEL);


template <typename ValueType, typename IndexType>
void remove_diagonal_elements(
    std::shared_ptr<const CudaExecutor> exec, const IndexType* row_ptrs,
    const IndexType* col_idxs,
    matrix::SparsityCsr<ValueType, IndexType>* matrix) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_REMOVE_DIAGONAL_ELEMENTS_KERNEL);


template <typename ValueType, typename IndexType>
void transpose(std::shared_ptr<const CudaExecutor> exec,
               const matrix::SparsityCsr<ValueType, IndexType>* orig,
               matrix::SparsityCsr<ValueType, IndexType>* trans)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_TRANSPOSE_KERNEL);


template <typename ValueType, typename IndexType>
void sort_by_column_index(std::shared_ptr<const CudaExecutor> exec,
                          matrix::SparsityCsr<ValueType, IndexType>* to_sort)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_SORT_BY_COLUMN_INDEX);


template <typename ValueType, typename IndexType>
void is_sorted_by_column_index(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::SparsityCsr<ValueType, IndexType>* to_check,
    bool* is_sorted) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_IS_SORTED_BY_COLUMN_INDEX);


}  // namespace sparsity_csr
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
