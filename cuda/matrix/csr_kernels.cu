#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2019, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/matrix/csr_kernels.hpp"


#include <algorithm>


#include <ginkgo/core/base/array.hpp>
#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/matrix/coo.hpp>
#include <ginkgo/core/matrix/dense.hpp>
#include <ginkgo/core/matrix/ell.hpp>
#include <ginkgo/core/matrix/hybrid.hpp>
#include <ginkgo/core/matrix/sellp.hpp>


#include "core/matrix/dense_kernels.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/pointer_mode_guard.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/atomic.cuh"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/prefix_sum.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/segment_scan.cuh"
#include "cuda/components/uninitialized_array.hpp"
#include "cuda/components/zero_array.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The Compressed sparse row matrix format namespace.
 *
 * @ingroup csr
 */
namespace csr {


constexpr int default_block_size = 512;
constexpr int warps_in_block = 4;
constexpr int spmv_block_size = warps_in_block * config::warp_size;
constexpr int classical_block_size = 64;
constexpr int wsize = config::warp_size;


/**
 * A compile-time list of the number items per threads for which spmv kernel
 * should be compiled.
 */
using compiled_kernels = syn::value_list<int, 3, 4, 6, 7, 8, 12, 14>;


#include "common/matrix/csr_kernels.hpp.inc"


namespace host_kernel {


template <int items_per_thread, typename ValueType, typename IndexType>
void merge_path_spmv(syn::value_list<int, items_per_thread>,
                     std::shared_ptr<const CudaExecutor> exec,
                     const matrix::Csr<ValueType, IndexType> *a,
                     const matrix::Dense<ValueType> *b,
                     matrix::Dense<ValueType> *c,
                     const matrix::Dense<ValueType> *alpha = nullptr,
                     const matrix::Dense<ValueType> *beta = nullptr)
{
    const IndexType total = a->get_size()[0] + a->get_num_stored_elements();
    const IndexType grid_num =
        ceildiv(total, spmv_block_size * items_per_thread);
    const dim3 grid(grid_num);
    const dim3 block(spmv_block_size);
    Array<IndexType> row_out(exec, grid_num);
    Array<ValueType> val_out(exec, grid_num);

    for (IndexType column_id = 0; column_id < b->get_size()[1]; column_id++) {
        if (alpha == nullptr && beta == nullptr) {
            const auto b_vals = b->get_const_values() + column_id;
            auto c_vals = c->get_values() + column_id;
            kernel::abstract_merge_path_spmv<items_per_thread>
                <<<grid, block, 0, 0>>>(
                    static_cast<IndexType>(a->get_size()[0]),
                    as_cuda_type(a->get_const_values()),
                    a->get_const_col_idxs(),
                    as_cuda_type(a->get_const_row_ptrs()),
                    as_cuda_type(a->get_const_srow()), as_cuda_type(b_vals),
                    b->get_stride(), as_cuda_type(c_vals), c->get_stride(),
                    as_cuda_type(row_out.get_data()),
                    as_cuda_type(val_out.get_data()));
            kernel::abstract_reduce<<<1, spmv_block_size>>>(
                grid_num, as_cuda_type(val_out.get_data()),
                as_cuda_type(row_out.get_data()), as_cuda_type(c_vals),
                c->get_stride());

        } else if (alpha != nullptr && beta != nullptr) {
            const auto b_vals = b->get_const_values() + column_id;
            auto c_vals = c->get_values() + column_id;
            kernel::abstract_merge_path_spmv<items_per_thread>
                <<<grid, block, 0, 0>>>(
                    static_cast<IndexType>(a->get_size()[0]),
                    as_cuda_type(alpha->get_const_values()),
                    as_cuda_type(a->get_const_values()),
                    a->get_const_col_idxs(),
                    as_cuda_type(a->get_const_row_ptrs()),
                    as_cuda_type(a->get_const_srow()), as_cuda_type(b_vals),
                    b->get_stride(), as_cuda_type(beta->get_const_values()),
                    as_cuda_type(c_vals), c->get_stride(),
                    as_cuda_type(row_out.get_data()),
                    as_cuda_type(val_out.get_data()));
            kernel::abstract_reduce<<<1, spmv_block_size>>>(
                grid_num, as_cuda_type(val_out.get_data()),
                as_cuda_type(row_out.get_data()),
                as_cuda_type(alpha->get_const_values()), as_cuda_type(c_vals),
                c->get_stride());
        } else {
            GKO_KERNEL_NOT_FOUND;
        }
    }
}

GKO_ENABLE_IMPLEMENTATION_SELECTION(select_merge_path_spmv, merge_path_spmv);


template <typename ValueType, typename IndexType>
int compute_items_per_thread(std::shared_ptr<const CudaExecutor> exec)
{
    const int version = exec->get_major_version()
                        << 4 + exec->get_minor_version();
    // The num_item is decided to make the occupancy 100%
    // TODO: Extend this list when new GPU is released
    //       Tune this parameter
    // 128 threads/block the number of items per threads
    // 3.0 3.5: 6
    // 3.7: 14
    // 5.0, 5.3, 6.0, 6.2: 8
    // 5.2, 6.1, 7.0: 12
    int num_item = 6;
    switch (version) {
    case 0x50:
    case 0x53:
    case 0x60:
    case 0x62:
        num_item = 8;
        break;
    case 0x52:
    case 0x61:
    case 0x70:
        num_item = 12;
        break;
    case 0x37:
        num_item = 14;
    }
    // Ensure that the following is satisfied:
    // sizeof(IndexType) + sizeof(ValueType)
    // <= items_per_thread * sizeof(IndexType)
    constexpr int minimal_num =
        ceildiv(sizeof(IndexType) + sizeof(ValueType), sizeof(IndexType));
    int items_per_thread = num_item * 4 / sizeof(IndexType);
    return std::max(minimal_num, items_per_thread);
}


}  // namespace host_kernel


template <typename ValueType, typename IndexType>
void spmv(std::shared_ptr<const CudaExecutor> exec,
          const matrix::Csr<ValueType, IndexType> *a,
          const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *c)
{
    if (a->get_strategy()->get_name() == "load_balance") {
        zero_array(c->get_num_stored_elements(), c->get_values());
        const IndexType nwarps = a->get_num_srow_elements();
        if (nwarps > 0) {
            const dim3 csr_block(config::warp_size, warps_in_block, 1);
            const dim3 csr_grid(ceildiv(nwarps, warps_in_block),
                                b->get_size()[1]);
            kernel::abstract_spmv<<<csr_grid, csr_block>>>(
                nwarps, static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()),
                as_cuda_type(b->get_stride()), as_cuda_type(c->get_values()),
                as_cuda_type(c->get_stride()));
        } else {
            GKO_NOT_SUPPORTED(nwarps);
        }
    } else if (a->get_strategy()->get_name() == "merge_path") {
        int items_per_thread =
            host_kernel::compute_items_per_thread<ValueType, IndexType>(exec);
        host_kernel::select_merge_path_spmv(
            compiled_kernels(),
            [&items_per_thread](int compiled_info) {
                return items_per_thread == compiled_info;
            },
            syn::value_list<int>(), syn::type_list<>(), exec, a, b, c);
    } else if (a->get_strategy()->get_name() == "classical") {
        const dim3 grid(ceildiv(a->get_size()[0], classical_block_size),
                        b->get_size()[1]);
        kernel::abstract_classical_spmv<<<grid, classical_block_size>>>(
            a->get_size()[0], as_cuda_type(a->get_const_values()),
            a->get_const_col_idxs(), as_cuda_type(a->get_const_row_ptrs()),
            as_cuda_type(b->get_const_values()), b->get_stride(),
            as_cuda_type(c->get_values()), c->get_stride());
    } else if (a->get_strategy()->get_name() == "sparselib" ||
               a->get_strategy()->get_name() == "cusparse") {
        if (cusparse::is_supported<ValueType, IndexType>::value) {
            // TODO: add implementation for int64 and multiple RHS
            auto handle = exec->get_cusparse_handle();
            auto descr = cusparse::create_mat_descr();
            {
                cusparse::pointer_mode_guard pm_guard(handle);
                auto row_ptrs = a->get_const_row_ptrs();
                auto col_idxs = a->get_const_col_idxs();
                auto alpha = one<ValueType>();
                auto beta = zero<ValueType>();
                if (b->get_stride() != 1 || c->get_stride() != 1) {
                    GKO_NOT_IMPLEMENTED;
                }
                cusparse::spmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               a->get_size()[0], a->get_size()[1],
                               a->get_num_stored_elements(), &alpha, descr,
                               a->get_const_values(), row_ptrs, col_idxs,
                               b->get_const_values(), &beta, c->get_values());
            }
            cusparse::destroy(descr);
        } else {
            GKO_NOT_IMPLEMENTED;
        }
    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_CSR_SPMV_KERNEL);


template <typename ValueType, typename IndexType>
void advanced_spmv(std::shared_ptr<const CudaExecutor> exec,
                   const matrix::Dense<ValueType> *alpha,
                   const matrix::Csr<ValueType, IndexType> *a,
                   const matrix::Dense<ValueType> *b,
                   const matrix::Dense<ValueType> *beta,
                   matrix::Dense<ValueType> *c)
{
    if (a->get_strategy()->get_name() == "load_balance") {
        dense::scale(exec, beta, c);

        const IndexType nwarps = a->get_num_srow_elements();

        if (nwarps > 0) {
            const dim3 csr_block(config::warp_size, warps_in_block, 1);
            const dim3 csr_grid(ceildiv(nwarps, warps_in_block),
                                b->get_size()[1]);
            kernel::abstract_spmv<<<csr_grid, csr_block>>>(
                nwarps, static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(alpha->get_const_values()),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()),
                as_cuda_type(b->get_stride()), as_cuda_type(c->get_values()),
                as_cuda_type(c->get_stride()));
        } else {
            GKO_NOT_SUPPORTED(nwarps);
        }
    } else if (a->get_strategy()->get_name() == "sparselib" ||
               a->get_strategy()->get_name() == "cusparse") {
        if (cusparse::is_supported<ValueType, IndexType>::value) {
            // TODO: add implementation for int64 and multiple RHS
            auto descr = cusparse::create_mat_descr();

            auto row_ptrs = a->get_const_row_ptrs();
            auto col_idxs = a->get_const_col_idxs();

            if (b->get_stride() != 1 || c->get_stride() != 1)
                GKO_NOT_IMPLEMENTED;

            cusparse::spmv(exec->get_cusparse_handle(),
                           HIPSPARSE_OPERATION_NON_TRANSPOSE, a->get_size()[0],
                           a->get_size()[1], a->get_num_stored_elements(),
                           alpha->get_const_values(), descr,
                           a->get_const_values(), row_ptrs, col_idxs,
                           b->get_const_values(), beta->get_const_values(),
                           c->get_values());

            cusparse::destroy(descr);
        } else {
            GKO_NOT_IMPLEMENTED;
        }
    } else if (a->get_strategy()->get_name() == "classical") {
        const dim3 grid(ceildiv(a->get_size()[0], classical_block_size),
                        b->get_size()[1]);
        kernel::abstract_classical_spmv<<<grid, classical_block_size>>>(
            a->get_size()[0], as_cuda_type(alpha->get_const_values()),
            as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
            as_cuda_type(a->get_const_row_ptrs()),
            as_cuda_type(b->get_const_values()), b->get_stride(),
            as_cuda_type(beta->get_const_values()),
            as_cuda_type(c->get_values()), c->get_stride());
    } else if (a->get_strategy()->get_name() == "merge_path") {
        int items_per_thread =
            host_kernel::compute_items_per_thread<ValueType, IndexType>(exec);
        host_kernel::select_merge_path_spmv(
            compiled_kernels(),
            [&items_per_thread](int compiled_info) {
                return items_per_thread == compiled_info;
            },
            syn::value_list<int>(), syn::type_list<>(), exec, a, b, c, alpha,
            beta);
    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_ADVANCED_SPMV_KERNEL);


template <typename ValueType, typename IndexType>
void spgemm(std::shared_ptr<const CudaExecutor> exec,
            const matrix::Csr<ValueType, IndexType> *a,
            const matrix::Csr<ValueType, IndexType> *b,
            const matrix::Csr<ValueType, IndexType> *c,
            Array<IndexType> &c_row_ptrs, Array<IndexType> &c_col_idxs,
            Array<ValueType> &c_vals) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_CSR_SPGEMM_KERNEL);


template <typename ValueType, typename IndexType>
void advanced_spgemm(std::shared_ptr<const CudaExecutor> exec,
                     const matrix::Dense<ValueType> *alpha,
                     const matrix::Csr<ValueType, IndexType> *a,
                     const matrix::Csr<ValueType, IndexType> *b,
                     const matrix::Dense<ValueType> *beta,
                     const matrix::Csr<ValueType, IndexType> *c,
                     Array<IndexType> &c_row_ptrs, Array<IndexType> &c_col_idxs,
                     Array<ValueType> &c_vals) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_ADVANCED_SPGEMM_KERNEL);


template <typename IndexType>
void convert_row_ptrs_to_idxs(std::shared_ptr<const CudaExecutor> exec,
                              const IndexType *ptrs, size_type num_rows,
                              IndexType *idxs)
{
    const auto grid_dim = ceildiv(num_rows, default_block_size);

    kernel::convert_row_ptrs_to_idxs<<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(ptrs), as_cuda_type(idxs));
}


template <typename ValueType, typename IndexType>
void convert_to_coo(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Coo<ValueType, IndexType> *result,
                    const matrix::Csr<ValueType, IndexType> *source)
{
    auto num_rows = result->get_size()[0];

    auto row_idxs = result->get_row_idxs();
    const auto source_row_ptrs = source->get_const_row_ptrs();

    convert_row_ptrs_to_idxs(exec, source_row_ptrs, num_rows, row_idxs);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONVERT_TO_COO_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_dense(std::shared_ptr<const CudaExecutor> exec,
                      matrix::Dense<ValueType> *result,
                      const matrix::Csr<ValueType, IndexType> *source)
{
    const auto num_rows = result->get_size()[0];
    const auto num_cols = result->get_size()[1];
    const auto stride = result->get_stride();
    const auto row_ptrs = source->get_const_row_ptrs();
    const auto col_idxs = source->get_const_col_idxs();
    const auto vals = source->get_const_values();

    const dim3 block_size(config::warp_size,
                          config::max_block_size / config::warp_size, 1);
    const dim3 init_grid_dim(ceildiv(stride, block_size.x),
                             ceildiv(num_rows, block_size.y), 1);
    kernel::initialize_zero_dense<<<init_grid_dim, block_size>>>(
        num_rows, num_cols, stride, as_cuda_type(result->get_values()));

    auto grid_dim = ceildiv(num_rows, default_block_size);
    kernel::fill_in_dense<<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(row_ptrs), as_cuda_type(col_idxs),
        as_cuda_type(vals), stride, as_cuda_type(result->get_values()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONVERT_TO_DENSE_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_sellp(std::shared_ptr<const CudaExecutor> exec,
                      matrix::Sellp<ValueType, IndexType> *result,
                      const matrix::Csr<ValueType, IndexType> *source)
{
    const auto num_rows = result->get_size()[0];
    const auto num_cols = result->get_size()[1];

    auto result_values = result->get_values();
    auto result_col_idxs = result->get_col_idxs();
    auto slice_lengths = result->get_slice_lengths();
    auto slice_sets = result->get_slice_sets();

    const auto slice_size = (result->get_slice_size() == 0)
                                ? matrix::default_slice_size
                                : result->get_slice_size();
    const auto stride_factor = (result->get_stride_factor() == 0)
                                   ? matrix::default_stride_factor
                                   : result->get_stride_factor();
    const int slice_num = ceildiv(num_rows, slice_size);

    const auto source_values = source->get_const_values();
    const auto source_row_ptrs = source->get_const_row_ptrs();
    const auto source_col_idxs = source->get_const_col_idxs();

    auto nnz_per_row = Array<size_type>(exec, num_rows);
    auto grid_dim = ceildiv(num_rows, default_block_size);

    kernel::calculate_nnz_per_row<<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(source_row_ptrs),
        as_cuda_type(nnz_per_row.get_data()));

    grid_dim = slice_num;

    kernel::calculate_slice_lengths<<<grid_dim, config::warp_size>>>(
        num_rows, slice_size, stride_factor,
        as_cuda_type(nnz_per_row.get_const_data()), as_cuda_type(slice_lengths),
        as_cuda_type(slice_sets));

    auto add_values =
        Array<size_type>(exec, ceildiv(slice_num + 1, default_block_size));
    grid_dim = ceildiv(slice_num + 1, default_block_size);

    start_prefix_sum<default_block_size><<<grid_dim, default_block_size>>>(
        slice_num + 1, as_cuda_type(slice_sets),
        as_cuda_type(add_values.get_data()));

    finalize_prefix_sum<default_block_size><<<grid_dim, default_block_size>>>(
        slice_num + 1, as_cuda_type(slice_sets),
        as_cuda_type(add_values.get_const_data()));

    grid_dim = ceildiv(num_rows, default_block_size);
    kernel::fill_in_sellp<<<grid_dim, default_block_size>>>(
        num_rows, slice_size, as_cuda_type(source_values),
        as_cuda_type(source_row_ptrs), as_cuda_type(source_col_idxs),
        as_cuda_type(slice_lengths), as_cuda_type(slice_sets),
        as_cuda_type(result_col_idxs), as_cuda_type(result_values));

    nnz_per_row.clear();
    add_values.clear();
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONVERT_TO_SELLP_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_ell(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Ell<ValueType, IndexType> *result,
                    const matrix::Csr<ValueType, IndexType> *source)
{
    const auto source_values = source->get_const_values();
    const auto source_row_ptrs = source->get_const_row_ptrs();
    const auto source_col_idxs = source->get_const_col_idxs();

    auto result_values = result->get_values();
    auto result_col_idxs = result->get_col_idxs();
    const auto stride = result->get_stride();
    const auto max_nnz_per_row = result->get_num_stored_elements_per_row();
    const auto num_rows = result->get_size()[0];
    const auto num_cols = result->get_size()[1];

    const auto init_grid_dim =
        ceildiv(max_nnz_per_row * num_rows, default_block_size);

    kernel::initialize_zero_ell<<<init_grid_dim, default_block_size>>>(
        max_nnz_per_row, stride, as_cuda_type(result_values),
        as_cuda_type(result_col_idxs));

    const auto grid_dim =
        ceildiv(num_rows * config::warp_size, default_block_size);

    kernel::fill_in_ell<<<grid_dim, default_block_size>>>(
        num_rows, stride, as_cuda_type(source_values),
        as_cuda_type(source_row_ptrs), as_cuda_type(source_col_idxs),
        as_cuda_type(result_values), as_cuda_type(result_col_idxs));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONVERT_TO_ELL_KERNEL);


template <typename ValueType, typename IndexType>
void calculate_total_cols(std::shared_ptr<const CudaExecutor> exec,
                          const matrix::Csr<ValueType, IndexType> *source,
                          size_type *result, size_type stride_factor,
                          size_type slice_size)
{
    const auto num_rows = source->get_size()[0];
    const auto slice_num = ceildiv(num_rows, slice_size);
    const auto row_ptrs = source->get_const_row_ptrs();

    auto nnz_per_row = Array<size_type>(exec, num_rows);
    auto grid_dim = ceildiv(num_rows, default_block_size);

    kernel::calculate_nnz_per_row<<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(row_ptrs), as_cuda_type(nnz_per_row.get_data()));

    grid_dim = ceildiv(slice_num * config::warp_size, default_block_size);
    auto max_nnz_per_slice = Array<size_type>(exec, slice_num);

    kernel::reduce_max_nnz_per_slice<<<grid_dim, default_block_size>>>(
        num_rows, slice_size, stride_factor,
        as_cuda_type(nnz_per_row.get_const_data()),
        as_cuda_type(max_nnz_per_slice.get_data()));

    grid_dim = ceildiv(slice_num, default_block_size);
    auto block_results = Array<size_type>(exec, grid_dim);

    kernel::reduce_total_cols<<<grid_dim, default_block_size>>>(
        slice_num, as_cuda_type(max_nnz_per_slice.get_const_data()),
        as_cuda_type(block_results.get_data()));

    auto d_result = Array<size_type>(exec, 1);

    kernel::reduce_total_cols<<<1, default_block_size>>>(
        grid_dim, as_cuda_type(block_results.get_const_data()),
        as_cuda_type(d_result.get_data()));

    exec->get_master()->get_mem_space()->copy_from(
        exec->get_mem_space().get(), 1, d_result.get_const_data(), result);

    block_results.clear();
    nnz_per_row.clear();
    max_nnz_per_slice.clear();
    d_result.clear();
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CALCULATE_TOTAL_COLS_KERNEL);


template <typename ValueType, typename IndexType>
void transpose(std::shared_ptr<const CudaExecutor> exec,
               matrix::Csr<ValueType, IndexType> *trans,
               const matrix::Csr<ValueType, IndexType> *orig)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

        cusparse::transpose(
            exec->get_cusparse_handle(), orig->get_size()[0],
            orig->get_size()[1], orig->get_num_stored_elements(),
            orig->get_const_values(), orig->get_const_row_ptrs(),
            orig->get_const_col_idxs(), trans->get_values(),
            trans->get_col_idxs(), trans->get_row_ptrs(), copyValues, idxBase);
    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_CSR_TRANSPOSE_KERNEL);


template <typename ValueType, typename IndexType>
void conj_transpose(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Csr<ValueType, IndexType> *trans,
                    const matrix::Csr<ValueType, IndexType> *orig)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        const dim3 block_size(default_block_size, 1, 1);
        const dim3 grid_size(
            ceildiv(trans->get_num_stored_elements(), block_size.x), 1, 1);

        hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

        cusparse::transpose(
            exec->get_cusparse_handle(), orig->get_size()[0],
            orig->get_size()[1], orig->get_num_stored_elements(),
            orig->get_const_values(), orig->get_const_row_ptrs(),
            orig->get_const_col_idxs(), trans->get_values(),
            trans->get_col_idxs(), trans->get_row_ptrs(), copyValues, idxBase);

        conjugate_kernel<<<grid_size, block_size, 0, 0>>>(
            trans->get_num_stored_elements(),
            as_cuda_type(trans->get_values()));
    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONJ_TRANSPOSE_KERNEL);


template <typename ValueType, typename IndexType>
void row_permute(std::shared_ptr<const CudaExecutor> exec,
                 const Array<IndexType> *permutation_indices,
                 matrix::Csr<ValueType, IndexType> *row_permuted,
                 const matrix::Csr<ValueType, IndexType> *orig)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_ROW_PERMUTE_KERNEL);


template <typename ValueType, typename IndexType>
void column_permute(std::shared_ptr<const CudaExecutor> exec,
                    const Array<IndexType> *permutation_indices,
                    matrix::Csr<ValueType, IndexType> *column_permuted,
                    const matrix::Csr<ValueType, IndexType> *orig)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_COLUMN_PERMUTE_KERNEL);


template <typename ValueType, typename IndexType>
void inverse_row_permute(std::shared_ptr<const CudaExecutor> exec,
                         const Array<IndexType> *permutation_indices,
                         matrix::Csr<ValueType, IndexType> *row_permuted,
                         const matrix::Csr<ValueType, IndexType> *orig)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_INVERSE_ROW_PERMUTE_KERNEL);


template <typename ValueType, typename IndexType>
void inverse_column_permute(std::shared_ptr<const CudaExecutor> exec,
                            const Array<IndexType> *permutation_indices,
                            matrix::Csr<ValueType, IndexType> *column_permuted,
                            const matrix::Csr<ValueType, IndexType> *orig)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_INVERSE_COLUMN_PERMUTE_KERNEL);


template <typename ValueType, typename IndexType>
void calculate_max_nnz_per_row(std::shared_ptr<const CudaExecutor> exec,
                               const matrix::Csr<ValueType, IndexType> *source,
                               size_type *result)
{
    const auto num_rows = source->get_size()[0];

    auto nnz_per_row = Array<size_type>(exec, num_rows);
    auto block_results = Array<size_type>(exec, default_block_size);
    auto d_result = Array<size_type>(exec, 1);

    const auto grid_dim = ceildiv(num_rows, default_block_size);
    kernel::calculate_nnz_per_row<<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(source->get_const_row_ptrs()),
        as_cuda_type(nnz_per_row.get_data()));

    const auto n = ceildiv(num_rows, default_block_size);
    const auto reduce_dim = n <= default_block_size ? n : default_block_size;
    kernel::reduce_max_nnz<<<reduce_dim, default_block_size>>>(
        num_rows, as_cuda_type(nnz_per_row.get_const_data()),
        as_cuda_type(block_results.get_data()));

    kernel::reduce_max_nnz<<<1, default_block_size>>>(
        reduce_dim, as_cuda_type(block_results.get_const_data()),
        as_cuda_type(d_result.get_data()));

    exec->get_master()->get_mem_space()->copy_from(
        exec->get_mem_space().get(), 1, d_result.get_const_data(), result);

    nnz_per_row.clear();
    block_results.clear();
    d_result.clear();
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CALCULATE_MAX_NNZ_PER_ROW_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_hybrid(std::shared_ptr<const CudaExecutor> exec,
                       matrix::Hybrid<ValueType, IndexType> *result,
                       const matrix::Csr<ValueType, IndexType> *source)
{
    auto ell_val = result->get_ell_values();
    auto ell_col = result->get_ell_col_idxs();
    auto coo_val = result->get_coo_values();
    auto coo_col = result->get_coo_col_idxs();
    auto coo_row = result->get_coo_row_idxs();
    const auto stride = result->get_ell_stride();
    const auto max_nnz_per_row = result->get_ell_num_stored_elements_per_row();
    const auto num_rows = result->get_size()[0];
    const auto coo_num_stored_elements = result->get_coo_num_stored_elements();
    auto grid_dim = ceildiv(max_nnz_per_row * num_rows, default_block_size);

    kernel::initialize_zero_ell<<<grid_dim, default_block_size>>>(
        max_nnz_per_row, stride, as_cuda_type(ell_val), as_cuda_type(ell_col));

    grid_dim = ceildiv(num_rows, default_block_size);
    auto coo_offset = Array<size_type>(exec, num_rows);
    kernel::calculate_hybrid_coo_row_nnz<<<grid_dim, default_block_size>>>(
        num_rows, max_nnz_per_row, as_cuda_type(source->get_const_row_ptrs()),
        as_cuda_type(coo_offset.get_data()));

    auto add_values =
        Array<size_type>(exec, ceildiv(num_rows, default_block_size));
    grid_dim = ceildiv(num_rows, default_block_size);
    start_prefix_sum<default_block_size><<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(coo_offset.get_data()),
        as_cuda_type(add_values.get_data()));
    finalize_prefix_sum<default_block_size><<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(coo_offset.get_data()),
        as_cuda_type(add_values.get_const_data()));

    grid_dim = ceildiv(num_rows * config::warp_size, default_block_size);
    kernel::fill_in_hybrid<<<grid_dim, default_block_size>>>(
        num_rows, stride, max_nnz_per_row,
        as_cuda_type(source->get_const_values()),
        as_cuda_type(source->get_const_row_ptrs()),
        as_cuda_type(source->get_const_col_idxs()),
        as_cuda_type(coo_offset.get_const_data()), as_cuda_type(ell_val),
        as_cuda_type(ell_col), as_cuda_type(coo_val), as_cuda_type(coo_col),
        as_cuda_type(coo_row));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONVERT_TO_HYBRID_KERNEL);


template <typename ValueType, typename IndexType>
void calculate_nonzeros_per_row(std::shared_ptr<const CudaExecutor> exec,
                                const matrix::Csr<ValueType, IndexType> *source,
                                Array<size_type> *result)
{
    const auto num_rows = source->get_size()[0];
    auto row_ptrs = source->get_const_row_ptrs();
    auto grid_dim = ceildiv(num_rows, default_block_size);

    kernel::calculate_nnz_per_row<<<grid_dim, default_block_size>>>(
        num_rows, as_cuda_type(row_ptrs), as_cuda_type(result->get_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CALCULATE_NONZEROS_PER_ROW_KERNEL);


template <typename ValueType, typename IndexType>
void sort_by_column_index(std::shared_ptr<const CudaExecutor> exec,
                          matrix::Csr<ValueType, IndexType> *to_sort)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_SORT_BY_COLUMN_INDEX);


template <typename ValueType, typename IndexType>
void is_sorted_by_column_index(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::Csr<ValueType, IndexType> *to_check,
    bool *is_sorted) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_IS_SORTED_BY_COLUMN_INDEX);


}  // namespace csr
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
