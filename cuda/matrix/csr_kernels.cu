#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2019

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/matrix/csr_kernels.hpp"


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>


#include "core/matrix/dense_kernels.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/atomic.cuh"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/synchronization.cuh"
#include "cuda/components/uninitialized_array.hpp"


namespace gko {
namespace kernels {
namespace cuda {
namespace csr {


constexpr int default_block_size = 512;
constexpr int warps_in_block = 4;
constexpr int spmv_block_size = warps_in_block * cuda_config::warp_size;
constexpr int classical_block_size = 64;
constexpr int wsize = cuda_config::warp_size;


namespace {


template <typename T>
__host__ __device__ __forceinline__ T ceildivT(T nom, T denom)
{
    return (nom + denom - 1ll) / denom;
}


template <typename ValueType, typename IndexType>
__device__ __forceinline__ bool segment_scan(
    const group::thread_block_tile<wsize> &group, const IndexType ind,
    ValueType *__restrict__ val)
{
    bool head = true;
#pragma unroll
    for (int i = 1; i < wsize; i <<= 1) {
        const IndexType add_ind = group.shfl_up(ind, i);
        ValueType add_val = zero<ValueType>();
        if (add_ind == ind && threadIdx.x >= i) {
            add_val = *val;
            if (i == 1) {
                head = false;
            }
        }
        add_val = group.shfl_down(add_val, i);
        if (threadIdx.x < wsize - i) {
            *val += add_val;
        }
    }
    return head;
}


template <typename ValueType, typename IndexType>
__device__ __forceinline__ bool block_segment_scan_reverse(
    const IndexType *__restrict__ ind, ValueType *__restrict__ val)
{
    bool last = true;
    const auto reg_ind = ind[threadIdx.x];
#pragma unroll
    for (int i = 1; i < spmv_block_size; i <<= 1) {
        if (i == 1 && threadIdx.x < spmv_block_size - 1 &&
            reg_ind == ind[threadIdx.x + 1]) {
            last = false;
        }
        auto temp = zero<ValueType>();
        if (threadIdx.x >= i && reg_ind == ind[threadIdx.x - i]) {
            temp = val[threadIdx.x - i];
        }
        __syncthreads();
        val[threadIdx.x] += temp;
        __syncthreads();
    }

    return last;
}


template <bool overflow, typename IndexType>
__device__ __forceinline__ void find_next_row(
    const IndexType num_rows, const IndexType data_size, const IndexType ind,
    IndexType *__restrict__ row, IndexType *__restrict__ row_end,
    const IndexType row_predict, const IndexType row_predict_end,
    const IndexType *__restrict__ row_ptr)
{
    if (!overflow || ind < data_size) {
        if (ind >= *row_end) {
            *row = row_predict;
            *row_end = row_predict_end;
            for (; ind >= *row_end; *row_end = row_ptr[++*row + 1])
                ;
        }

    } else {
        *row = num_rows - 1;
        *row_end = data_size;
    }
}


template <typename ValueType, typename IndexType, typename Closure>
__device__ __forceinline__ void warp_atomic_add(
    const group::thread_block_tile<wsize> &group, bool force_write,
    ValueType *__restrict__ val, IndexType ind, ValueType *__restrict__ out,
    Closure scale)
{
    // do a local scan to avoid atomic collisions
    const bool need_write = segment_scan(group, ind, val);
    if (need_write && force_write) {
        atomic_add(out + ind, scale(*val));
    }
    if (!need_write || force_write) {
        *val = zero<ValueType>();
    }
}


template <bool last, typename ValueType, typename IndexType, typename Closure>
__device__ __forceinline__ void process_window(
    const group::thread_block_tile<wsize> &group, const IndexType num_rows,
    const IndexType data_size, const IndexType ind, IndexType *__restrict__ row,
    IndexType *__restrict__ row_end, IndexType *__restrict__ nrow,
    IndexType *__restrict__ nrow_end, ValueType *__restrict__ temp_val,
    const ValueType *__restrict__ val, const IndexType *__restrict__ col_idxs,
    const IndexType *__restrict__ row_ptrs, const ValueType *__restrict__ b,
    ValueType *__restrict__ c, Closure scale)
{
    const IndexType curr_row = *row;
    find_next_row<last>(num_rows, data_size, ind, row, row_end, *nrow,
                        *nrow_end, row_ptrs);
    // segmented scan
    if (group.any(curr_row != *row)) {
        warp_atomic_add(group, curr_row != *row, temp_val, curr_row, c, scale);
        *nrow = group.shfl(*row, wsize - 1);
        *nrow_end = group.shfl(*row_end, wsize - 1);
    }

    if (!last || ind < data_size) {
        const auto col = col_idxs[ind];
        *temp_val += val[ind] * b[col];
    }
}


template <typename IndexType>
__device__ __forceinline__ IndexType get_warp_start_idx(
    const IndexType nwarps, const IndexType nnz, const IndexType warp_idx)
{
    const long long cache_lines = ceildivT<IndexType>(nnz, wsize);
    return (warp_idx * cache_lines / nwarps) * wsize;
}


template <typename ValueType, typename IndexType, typename Closure>
__device__ __forceinline__ void spmv_kernel(
    const IndexType nwarps, const IndexType num_rows,
    const ValueType *__restrict__ val, const IndexType *__restrict__ col_idxs,
    const IndexType *__restrict__ row_ptrs, const IndexType *__restrict__ srow,
    const ValueType *__restrict__ b, ValueType *__restrict__ c, Closure scale)
{
    const IndexType warp_idx = blockIdx.x * warps_in_block + threadIdx.y;
    if (warp_idx >= nwarps) {
        return;
    }
    const IndexType data_size = row_ptrs[num_rows];
    const IndexType start = get_warp_start_idx(nwarps, data_size, warp_idx);
    const IndexType end =
        min(get_warp_start_idx(nwarps, data_size, warp_idx + 1),
            ceildivT<IndexType>(data_size, wsize) * wsize);
    auto row = srow[warp_idx];
    auto row_end = row_ptrs[row + 1];
    auto nrow = row;
    auto nrow_end = row_end;
    ValueType temp_val = zero<ValueType>();
    IndexType ind = start + threadIdx.x;
    find_next_row<true>(num_rows, data_size, ind, &row, &row_end, nrow,
                        nrow_end, row_ptrs);
    const IndexType ind_end = end - wsize;
    const auto tile_block =
        group::tiled_partition<wsize>(group::this_thread_block());
    for (; ind < ind_end; ind += wsize) {
        process_window<false>(tile_block, num_rows, data_size, ind, &row,
                              &row_end, &nrow, &nrow_end, &temp_val, val,
                              col_idxs, row_ptrs, b, c, scale);
    }
    process_window<true>(tile_block, num_rows, data_size, ind, &row, &row_end,
                         &nrow, &nrow_end, &temp_val, val, col_idxs, row_ptrs,
                         b, c, scale);
    warp_atomic_add(tile_block, true, &temp_val, row, c, scale);
}


template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(spmv_block_size) void abstract_spmv(
    const IndexType nwarps, const IndexType num_rows,
    const ValueType *__restrict__ val, const IndexType *__restrict__ col_idxs,
    const IndexType *__restrict__ row_ptrs, const IndexType *__restrict__ srow,
    const ValueType *__restrict__ b, ValueType *__restrict__ c)
{
    spmv_kernel(nwarps, num_rows, val, col_idxs, row_ptrs, srow, b, c,
                [](const ValueType &x) { return x; });
}


template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(spmv_block_size) void abstract_spmv(
    const IndexType nwarps, const IndexType num_rows,
    const ValueType *__restrict__ alpha, const ValueType *__restrict__ val,
    const IndexType *__restrict__ col_idxs,
    const IndexType *__restrict__ row_ptrs, const IndexType *__restrict__ srow,
    const ValueType *__restrict__ b, ValueType *__restrict__ c)
{
    ValueType scale_factor = alpha[0];
    spmv_kernel(
        nwarps, num_rows, val, col_idxs, row_ptrs, srow, b, c,
        [&scale_factor](const ValueType &x) { return scale_factor * x; });
}


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void set_zero(
    const size_type nnz, ValueType *__restrict__ val)
{
    const auto ind =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    if (ind < nnz) {
        val[ind] = zero<ValueType>();
    }
}


template <typename IndexType>
__forceinline__ __device__ void merge_path_search(
    const IndexType diagonal, const IndexType a_len, const IndexType b_len,
    const IndexType *__restrict__ a, const IndexType offset_b,
    IndexType *__restrict__ x, IndexType *__restrict__ y)
{
    auto x_min = max(diagonal - b_len, zero<IndexType>());
    auto x_max = min(diagonal, a_len);
    while (x_min < x_max) {
        auto pivot = (x_min + x_max) >> 1;
        if (a[pivot] <= offset_b + diagonal - pivot - 1) {
            x_min = pivot + 1;
        } else {
            x_max = pivot;
        }
    }

    *x = min(x_min, a_len);
    *y = diagonal - x_min;
}


template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(spmv_block_size) void reduce(
    const IndexType nwarps, const ValueType *__restrict__ last_val,
    const IndexType *__restrict__ last_row, ValueType *__restrict__ c,
    const size_type c_stride)
{
    const IndexType cache_lines = ceildivT<IndexType>(nwarps, spmv_block_size);
    const IndexType tid = threadIdx.x;
    const IndexType start = min(tid * cache_lines, nwarps);
    const IndexType end = min((tid + 1) * cache_lines, nwarps);
    ValueType value = zero<ValueType>();
    IndexType row = last_row[nwarps - 1];
    if (start < nwarps) {
        value = last_val[start];
        row = last_row[start];
        for (IndexType i = start + 1; i < end; i++) {
            if (last_row[i] != row) {
                c[row] += value;
                row = last_row[i];
                value = last_val[i];
            } else {
                value += last_val[i];
            }
        }
    }
    __shared__ UninitializedArray<IndexType, spmv_block_size> tmp_ind;
    __shared__ UninitializedArray<ValueType, spmv_block_size> tmp_val;
    tmp_val[threadIdx.x] = value;
    tmp_ind[threadIdx.x] = row;
    __syncthreads();
    bool last = block_segment_scan_reverse(static_cast<IndexType *>(tmp_ind),
                                           static_cast<ValueType *>(tmp_val));
    __syncthreads();
    if (last) {
        c[row] += tmp_val[threadIdx.x];
    }
}


template <int items_per_thread, typename ValueType, typename IndexType>
__global__ __launch_bounds__(spmv_block_size) void merge_path_spmv(
    const IndexType num_rows, const ValueType *__restrict__ val,
    const IndexType *__restrict__ col_idxs,
    const IndexType *__restrict__ row_ptrs, const IndexType *__restrict__ srow,
    const ValueType *__restrict__ b, const size_type b_stride,
    ValueType *__restrict__ c, const size_type c_stride,
    IndexType *__restrict__ row_out, ValueType *__restrict__ val_out)
{
    const auto *row_end_ptrs = row_ptrs + 1;
    const auto nnz = row_ptrs[num_rows];
    const IndexType num_merge_items = num_rows + nnz;
    const auto block_items = spmv_block_size * items_per_thread;
    __shared__ IndexType shared_row_ptrs[block_items];
    const IndexType diagonal =
        min(static_cast<IndexType>(block_items * blockIdx.x), num_merge_items);
    const IndexType diagonal_end = min(diagonal + block_items, num_merge_items);
    IndexType block_start_x;
    IndexType block_start_y;
    IndexType end_x;
    IndexType end_y;
    merge_path_search(diagonal, num_rows, nnz, row_end_ptrs, zero<IndexType>(),
                      &block_start_x, &block_start_y);
    merge_path_search(diagonal_end, num_rows, nnz, row_end_ptrs,
                      zero<IndexType>(), &end_x, &end_y);
    const IndexType block_num_rows = end_x - block_start_x;
    const IndexType block_num_nonzeros = end_y - block_start_y;
    for (int i = threadIdx.x;
         i < block_num_rows && block_start_x + i < num_rows;
         i += spmv_block_size) {
        shared_row_ptrs[i] = row_end_ptrs[block_start_x + i];
    }
    __syncthreads();

    IndexType start_x;
    IndexType start_y;
    merge_path_search(static_cast<IndexType>(items_per_thread * threadIdx.x),
                      block_num_rows, block_num_nonzeros, shared_row_ptrs,
                      block_start_y, &start_x, &start_y);

    ValueType value = zero<ValueType>();
#pragma unroll
    for (IndexType i = 0; i < items_per_thread; i++) {
        const IndexType ind = block_start_y + start_y;
        const IndexType row_i = block_start_x + start_x;
        if (row_i < num_rows) {
            if (start_x == block_num_rows || ind < shared_row_ptrs[start_x]) {
                value += val[ind] * b[col_idxs[ind]];
                start_y++;
            } else {
                c[row_i] = value;
                value = zero<ValueType>();
                start_x++;
            }
        }
    }
    __syncthreads();
    IndexType *tmp_ind = shared_row_ptrs;
    ValueType *tmp_val =
        reinterpret_cast<ValueType *>(shared_row_ptrs + spmv_block_size);
    tmp_val[threadIdx.x] = value;
    tmp_ind[threadIdx.x] = block_start_x + start_x;
    __syncthreads();
    bool last = block_segment_scan_reverse(static_cast<IndexType *>(tmp_ind),
                                           static_cast<ValueType *>(tmp_val));
    if (threadIdx.x == spmv_block_size - 1) {
        row_out[blockIdx.x] = min(end_x, num_rows - 1);
        val_out[blockIdx.x] = tmp_val[threadIdx.x];
    } else if (last) {
        c[block_start_x + start_x] += tmp_val[threadIdx.x];
    }
}


template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(64) void classical_spmv(
    const size_type num_rows, const ValueType *__restrict__ val,
    const IndexType *__restrict__ col_idxs,
    const IndexType *__restrict__ row_ptrs, const ValueType *__restrict__ b,
    const size_type b_stride, ValueType *__restrict__ c,
    const size_type c_stride)
{
    const auto tid =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    if (tid >= num_rows) {
        return;
    }
    const auto column_id = blockIdx.y;
    const auto ind_end = row_ptrs[tid + 1];
    ValueType temp_value = zero<ValueType>();
    for (auto ind = row_ptrs[tid]; ind < ind_end; ind++) {
        temp_value += val[ind] * b[col_idxs[ind] * b_stride + column_id];
    }
    c[tid * c_stride + column_id] = temp_value;
}


}  // namespace


template <typename ValueType, typename IndexType>
void spmv(std::shared_ptr<const CudaExecutor> exec,
          const matrix::Csr<ValueType, IndexType> *a,
          const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *c)
{
    if (a->get_strategy()->get_name() == "load_balance") {
        ASSERT_NO_CUDA_ERRORS(
            hipMemset(c->get_values(), 0,
                       c->get_num_stored_elements() * sizeof(ValueType)));
        const IndexType nwarps = a->get_num_srow_elements();
        if (nwarps > 0) {
            const dim3 csr_block(cuda_config::warp_size, warps_in_block, 1);
            const dim3 csr_grid(ceildiv(nwarps, warps_in_block));
            abstract_spmv<<<csr_grid, csr_block>>>(
                nwarps, static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()),
                as_cuda_type(c->get_values()));
        }
    } else if (a->get_strategy()->get_name() == "merge_path") {
        const int version = exec->get_major_version()
                            << 4 + exec->get_minor_version();
        // 128 threads/block the number of items per threads
        // 3.0 3.5: 6
        // 3.7: 14
        // 5.0, 5.3, 6.0, 6.2: 8
        // 5.2, 6.1, 7.0: 12
        int num_item = 6;
        switch (version) {
        case 0x50:
        case 0x53:
        case 0x60:
        case 0x62:
            num_item = 8;
            break;
        case 0x52:
        case 0x61:
        case 0x70:
            num_item = 12;
            break;
        case 0x37:
            num_item = 14;
        }
        // The calculation is based on size(IndexType) = 4
        constexpr int index_scale = sizeof(IndexType) / 4;
        const int items_per_thread = num_item / index_scale;

        const IndexType total = a->get_size()[0] + a->get_num_stored_elements();
        const IndexType grid_num =
            ceildiv(total, spmv_block_size * items_per_thread);
        const dim3 grid(grid_num);
        const dim3 block(spmv_block_size);
        Array<IndexType> row_out(exec, grid_num);
        Array<ValueType> val_out(exec, grid_num);
        if (num_item == 6) {
            merge_path_spmv<6 / index_scale><<<grid, block>>>(
                static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()), b->get_stride(),
                as_cuda_type(c->get_values()), c->get_stride(),
                as_cuda_type(row_out.get_data()),
                as_cuda_type(val_out.get_data()));
        } else if (num_item == 8) {
            merge_path_spmv<8 / index_scale><<<grid, block>>>(
                static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()), b->get_stride(),
                as_cuda_type(c->get_values()), c->get_stride(),
                as_cuda_type(row_out.get_data()),
                as_cuda_type(val_out.get_data()));
        } else if (num_item == 12) {
            merge_path_spmv<12 / index_scale><<<grid, block>>>(
                static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()), b->get_stride(),
                as_cuda_type(c->get_values()), c->get_stride(),
                as_cuda_type(row_out.get_data()),
                as_cuda_type(val_out.get_data()));
        } else if (num_item == 14) {
            merge_path_spmv<14 / index_scale><<<grid, block>>>(
                static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()), b->get_stride(),
                as_cuda_type(c->get_values()), c->get_stride(),
                as_cuda_type(row_out.get_data()),
                as_cuda_type(val_out.get_data()));
        }

        reduce<<<1, spmv_block_size>>>(
            grid_num, as_cuda_type(val_out.get_data()),
            as_cuda_type(row_out.get_data()), as_cuda_type(c->get_values()),
            c->get_stride());
    } else if (a->get_strategy()->get_name() == "classical") {
        classical_spmv<<<ceildiv(a->get_size()[0], classical_block_size),
                         classical_block_size>>>(
            a->get_size()[0], as_cuda_type(a->get_const_values()),
            a->get_const_col_idxs(), as_cuda_type(a->get_const_row_ptrs()),
            as_cuda_type(b->get_const_values()), b->get_stride(),
            as_cuda_type(c->get_values()), c->get_stride());
    } else if (a->get_strategy()->get_name() == "cusparse") {
        if (cusparse::is_supported<ValueType, IndexType>::value) {
            // TODO: add implementation for int64 and multiple RHS
            auto handle = exec->get_cusparse_handle();
            auto descr = cusparse::create_mat_descr();
            ASSERT_NO_CUSPARSE_ERRORS(
                hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

            auto row_ptrs = a->get_const_row_ptrs();
            auto col_idxs = a->get_const_col_idxs();
            auto alpha = one<ValueType>();
            auto beta = zero<ValueType>();
            if (b->get_stride() != 1 || c->get_stride() != 1) NOT_IMPLEMENTED;

            cusparse::spmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                           a->get_size()[0], a->get_size()[1],
                           a->get_num_stored_elements(), &alpha, descr,
                           a->get_const_values(), row_ptrs, col_idxs,
                           b->get_const_values(), &beta, c->get_values());

            ASSERT_NO_CUSPARSE_ERRORS(
                hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));

            cusparse::destroy(descr);
        } else {
            // use classical implementation
            classical_spmv<<<ceildiv(a->get_size()[0], classical_block_size),
                             classical_block_size>>>(
                a->get_size()[0], as_cuda_type(a->get_const_values()),
                a->get_const_col_idxs(), as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(b->get_const_values()), b->get_stride(),
                as_cuda_type(c->get_values()), c->get_stride());
        }
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_CSR_SPMV_KERNEL);


template <typename ValueType, typename IndexType>
void advanced_spmv(std::shared_ptr<const CudaExecutor> exec,
                   const matrix::Dense<ValueType> *alpha,
                   const matrix::Csr<ValueType, IndexType> *a,
                   const matrix::Dense<ValueType> *b,
                   const matrix::Dense<ValueType> *beta,
                   matrix::Dense<ValueType> *c)
{
    if (a->get_strategy()->get_name() == "load_balance") {
        dense::scale(exec, beta, c);

        const IndexType nwarps = a->get_num_srow_elements();

        if (nwarps > 0) {
            const dim3 csr_block(cuda_config::warp_size, warps_in_block, 1);
            const dim3 csr_grid(ceildiv(nwarps, warps_in_block));
            abstract_spmv<<<csr_grid, csr_block>>>(
                nwarps, static_cast<IndexType>(a->get_size()[0]),
                as_cuda_type(alpha->get_const_values()),
                as_cuda_type(a->get_const_values()), a->get_const_col_idxs(),
                as_cuda_type(a->get_const_row_ptrs()),
                as_cuda_type(a->get_const_srow()),
                as_cuda_type(b->get_const_values()),
                as_cuda_type(c->get_values()));
        }
    } else if (a->get_strategy()->get_name() == "cusparse") {
        if (cusparse::is_supported<ValueType, IndexType>::value) {
            // TODO: add implementation for int64 and multiple RHS
            auto descr = cusparse::create_mat_descr();

            auto row_ptrs = a->get_const_row_ptrs();
            auto col_idxs = a->get_const_col_idxs();

            if (b->get_stride() != 1 || c->get_stride() != 1) NOT_IMPLEMENTED;

            cusparse::spmv(exec->get_cusparse_handle(),
                           HIPSPARSE_OPERATION_NON_TRANSPOSE, a->get_size()[0],
                           a->get_size()[1], a->get_num_stored_elements(),
                           alpha->get_const_values(), descr,
                           a->get_const_values(), row_ptrs, col_idxs,
                           b->get_const_values(), beta->get_const_values(),
                           c->get_values());

            cusparse::destroy(descr);
        } else {
            NOT_IMPLEMENTED;
        }
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_ADVANCED_SPMV_KERNEL);


template <typename IndexType>
void convert_row_ptrs_to_idxs(std::shared_ptr<const CudaExecutor> exec,
                              const IndexType *ptrs, size_type num_rows,
                              IndexType *idxs) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_INDEX_TYPE(
    GKO_DECLARE_CSR_CONVERT_ROW_PTRS_TO_IDXS_KERNEL);


template <typename ValueType, typename IndexType>
void convert_to_dense(
    std::shared_ptr<const CudaExecutor> exec, matrix::Dense<ValueType> *result,
    const matrix::Csr<ValueType, IndexType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONVERT_TO_DENSE_KERNEL);


template <typename ValueType, typename IndexType>
void move_to_dense(std::shared_ptr<const CudaExecutor> exec,
                   matrix::Dense<ValueType> *result,
                   matrix::Csr<ValueType, IndexType> *source) NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_MOVE_TO_DENSE_KERNEL);


template <typename ValueType, typename IndexType>
void transpose(std::shared_ptr<const CudaExecutor> exec,
               matrix::Csr<ValueType, IndexType> *trans,
               const matrix::Csr<ValueType, IndexType> *orig)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

        cusparse::transpose(
            exec->get_cusparse_handle(), orig->get_size()[0],
            orig->get_size()[1], orig->get_num_stored_elements(),
            orig->get_const_values(), orig->get_const_row_ptrs(),
            orig->get_const_col_idxs(), trans->get_values(),
            trans->get_col_idxs(), trans->get_row_ptrs(), copyValues, idxBase);
    } else {
        NOT_IMPLEMENTED;
    }
}


GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_CSR_TRANSPOSE_KERNEL);


namespace {


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void conjugate_kernel(
    size_type num_nonzeros, ValueType *__restrict__ val)
{
    const auto tidx =
        static_cast<size_type>(blockIdx.x) * default_block_size + threadIdx.x;

    if (tidx < num_nonzeros) {
        val[tidx] = conj(val[tidx]);
    }
}


}  //  namespace


template <typename ValueType, typename IndexType>
void conj_transpose(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Csr<ValueType, IndexType> *trans,
                    const matrix::Csr<ValueType, IndexType> *orig)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        const dim3 block_size(default_block_size, 1, 1);
        const dim3 grid_size(
            ceildiv(trans->get_num_stored_elements(), block_size.x), 1, 1);

        hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

        cusparse::transpose(
            exec->get_cusparse_handle(), orig->get_size()[0],
            orig->get_size()[1], orig->get_num_stored_elements(),
            orig->get_const_values(), orig->get_const_row_ptrs(),
            orig->get_const_col_idxs(), trans->get_values(),
            trans->get_col_idxs(), trans->get_row_ptrs(), copyValues, idxBase);

        conjugate_kernel<<<grid_size, block_size, 0, 0>>>(
            trans->get_num_stored_elements(),
            as_cuda_type(trans->get_values()));
    } else {
        NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_CSR_CONJ_TRANSPOSE_KERNEL);


}  // namespace csr
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
