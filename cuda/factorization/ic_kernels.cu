/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2022, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/factorization/ic_kernels.hpp"


#include <ginkgo/core/base/array.hpp>


#include "cuda/base/cusparse_bindings.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The ic factorization namespace.
 *
 * @ingroup factor
 */
namespace ic_factorization {


template <typename ValueType, typename IndexType>
void compute(std::shared_ptr<const DefaultExecutor> exec,
             matrix::Csr<ValueType, IndexType>* m)
{
    const auto id = exec->get_device_id();
    auto handle = exec->get_cusparse_handle();
    auto desc = cusparse::create_mat_descr();
    auto info = cusparse::create_ic0_info();

    // get buffer size for IC
    IndexType num_rows = m->get_size()[0];
    IndexType nnz = m->get_num_stored_elements();
    size_type buffer_size{};
    cusparse::ic0_buffer_size(handle, num_rows, nnz, desc,
                              m->get_const_values(), m->get_const_row_ptrs(),
                              m->get_const_col_idxs(), info, buffer_size);

    array<char> buffer{exec, buffer_size};

    // set up IC(0)
    cusparse::ic0_analysis(handle, num_rows, nnz, desc, m->get_const_values(),
                           m->get_const_row_ptrs(), m->get_const_col_idxs(),
                           info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL,
                           buffer.get_data());

    cusparse::ic0(handle, num_rows, nnz, desc, m->get_values(),
                  m->get_const_row_ptrs(), m->get_const_col_idxs(), info,
                  HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer.get_data());

    // CUDA 11.4 has a use-after-free bug on Turing
#if (CUDA_VERSION >= 11040)
    exec->synchronize();
#endif

    cusparse::destroy(info);
    cusparse::destroy(desc);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_IC_COMPUTE_KERNEL);


}  // namespace ic_factorization
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
