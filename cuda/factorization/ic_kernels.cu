// SPDX-FileCopyrightText: 2017 - 2024 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/factorization/ic_kernels.hpp"


#include <ginkgo/core/base/array.hpp>


#include "cuda/base/cusparse_bindings.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The ic factorization namespace.
 *
 * @ingroup factor
 */
namespace ic_factorization {


template <typename ValueType, typename IndexType>
void compute(std::shared_ptr<const DefaultExecutor> exec,
             matrix::Csr<ValueType, IndexType>* m)
{
    const auto id = exec->get_device_id();
    auto handle = exec->get_cusparse_handle();
    auto desc = cusparse::create_mat_descr();
    auto info = cusparse::create_ic0_info();

    // get buffer size for IC
    IndexType num_rows = m->get_size()[0];
    IndexType nnz = m->get_num_stored_elements();
    size_type buffer_size{};
    cusparse::ic0_buffer_size(handle, num_rows, nnz, desc,
                              m->get_const_values(), m->get_const_row_ptrs(),
                              m->get_const_col_idxs(), info, buffer_size);

    array<char> buffer{exec, buffer_size};

    // set up IC(0)
    cusparse::ic0_analysis(handle, num_rows, nnz, desc, m->get_const_values(),
                           m->get_const_row_ptrs(), m->get_const_col_idxs(),
                           info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL,
                           buffer.get_data());

    cusparse::ic0(handle, num_rows, nnz, desc, m->get_values(),
                  m->get_const_row_ptrs(), m->get_const_col_idxs(), info,
                  HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer.get_data());

    // CUDA 11.4 has a use-after-free bug on Turing
#if (CUDA_VERSION >= 11040)
    exec->synchronize();
#endif

    cusparse::destroy(info);
    cusparse::destroy(desc);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_IC_COMPUTE_KERNEL);


}  // namespace ic_factorization
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
