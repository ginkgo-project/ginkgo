// SPDX-FileCopyrightText: 2017-2023 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/factorization/par_ilut_kernels.hpp"


#include <ginkgo/core/base/array.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/matrix/coo.hpp>
#include <ginkgo/core/matrix/csr.hpp>
#include <ginkgo/core/matrix/dense.hpp>


#include "core/components/prefix_sum_kernels.hpp"
#include "core/matrix/coo_builder.hpp"
#include "core/matrix/csr_builder.hpp"
#include "core/matrix/csr_kernels.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/math.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/intrinsics.cuh"
#include "cuda/components/merging.cuh"
#include "cuda/components/prefix_sum.cuh"
#include "cuda/components/searching.cuh"
#include "cuda/components/thread_ids.cuh"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The parallel ILUT factorization namespace.
 *
 * @ingroup factor
 */
namespace par_ilut_factorization {


constexpr int default_block_size = 512;


// subwarp sizes for add_candidates kernels
using compiled_kernels =
    syn::value_list<int, 1, 2, 4, 8, 16, 32, config::warp_size>;


#include "common/cuda_hip/factorization/par_ilut_spgeam_kernels.hpp.inc"


namespace {


template <int subwarp_size, typename ValueType, typename IndexType>
void add_candidates(syn::value_list<int, subwarp_size>,
                    std::shared_ptr<const DefaultExecutor> exec,
                    const matrix::Csr<ValueType, IndexType>* lu,
                    const matrix::Csr<ValueType, IndexType>* a,
                    const matrix::Csr<ValueType, IndexType>* l,
                    const matrix::Csr<ValueType, IndexType>* u,
                    matrix::Csr<ValueType, IndexType>* l_new,
                    matrix::Csr<ValueType, IndexType>* u_new)
{
    auto num_rows = static_cast<IndexType>(lu->get_size()[0]);
    auto subwarps_per_block = default_block_size / subwarp_size;
    auto num_blocks = ceildiv(num_rows, subwarps_per_block);
    matrix::CsrBuilder<ValueType, IndexType> l_new_builder(l_new);
    matrix::CsrBuilder<ValueType, IndexType> u_new_builder(u_new);
    auto lu_row_ptrs = lu->get_const_row_ptrs();
    auto lu_col_idxs = lu->get_const_col_idxs();
    auto lu_vals = lu->get_const_values();
    auto a_row_ptrs = a->get_const_row_ptrs();
    auto a_col_idxs = a->get_const_col_idxs();
    auto a_vals = a->get_const_values();
    auto l_row_ptrs = l->get_const_row_ptrs();
    auto l_col_idxs = l->get_const_col_idxs();
    auto l_vals = l->get_const_values();
    auto u_row_ptrs = u->get_const_row_ptrs();
    auto u_col_idxs = u->get_const_col_idxs();
    auto u_vals = u->get_const_values();
    auto l_new_row_ptrs = l_new->get_row_ptrs();
    auto u_new_row_ptrs = u_new->get_row_ptrs();
    // count non-zeros per row
    if (num_blocks > 0) {
        kernel::tri_spgeam_nnz<subwarp_size>
            <<<num_blocks, default_block_size, 0, exec->get_stream()>>>(
                lu_row_ptrs, lu_col_idxs, a_row_ptrs, a_col_idxs,
                l_new_row_ptrs, u_new_row_ptrs, num_rows);
    }

    // build row ptrs
    components::prefix_sum_nonnegative(exec, l_new_row_ptrs, num_rows + 1);
    components::prefix_sum_nonnegative(exec, u_new_row_ptrs, num_rows + 1);

    // resize output arrays
    auto l_new_nnz = exec->copy_val_to_host(l_new_row_ptrs + num_rows);
    auto u_new_nnz = exec->copy_val_to_host(u_new_row_ptrs + num_rows);
    l_new_builder.get_col_idx_array().resize_and_reset(l_new_nnz);
    l_new_builder.get_value_array().resize_and_reset(l_new_nnz);
    u_new_builder.get_col_idx_array().resize_and_reset(u_new_nnz);
    u_new_builder.get_value_array().resize_and_reset(u_new_nnz);

    auto l_new_col_idxs = l_new->get_col_idxs();
    auto l_new_vals = l_new->get_values();
    auto u_new_col_idxs = u_new->get_col_idxs();
    auto u_new_vals = u_new->get_values();

    // fill columns and values
    if (num_blocks > 0) {
        kernel::tri_spgeam_init<subwarp_size>
            <<<num_blocks, default_block_size, 0, exec->get_stream()>>>(
                lu_row_ptrs, lu_col_idxs, as_device_type(lu_vals), a_row_ptrs,
                a_col_idxs, as_device_type(a_vals), l_row_ptrs, l_col_idxs,
                as_device_type(l_vals), u_row_ptrs, u_col_idxs,
                as_device_type(u_vals), l_new_row_ptrs, l_new_col_idxs,
                as_device_type(l_new_vals), u_new_row_ptrs, u_new_col_idxs,
                as_device_type(u_new_vals), num_rows);
    }
}


GKO_ENABLE_IMPLEMENTATION_SELECTION(select_add_candidates, add_candidates);


}  // namespace


template <typename ValueType, typename IndexType>
void add_candidates(std::shared_ptr<const DefaultExecutor> exec,
                    const matrix::Csr<ValueType, IndexType>* lu,
                    const matrix::Csr<ValueType, IndexType>* a,
                    const matrix::Csr<ValueType, IndexType>* l,
                    const matrix::Csr<ValueType, IndexType>* u,
                    matrix::Csr<ValueType, IndexType>* l_new,
                    matrix::Csr<ValueType, IndexType>* u_new)
{
    auto num_rows = a->get_size()[0];
    auto total_nnz =
        lu->get_num_stored_elements() + a->get_num_stored_elements();
    auto total_nnz_per_row = total_nnz / num_rows;
    select_add_candidates(
        compiled_kernels(),
        [&](int compiled_subwarp_size) {
            return total_nnz_per_row <= compiled_subwarp_size ||
                   compiled_subwarp_size == config::warp_size;
        },
        syn::value_list<int>(), syn::type_list<>(), exec, lu, a, l, u, l_new,
        u_new);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_PAR_ILUT_ADD_CANDIDATES_KERNEL);


}  // namespace par_ilut_factorization
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
