// SPDX-FileCopyrightText: 2017-2023 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/multigrid/pgm_kernels.hpp"


#include <memory>


#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/multigrid/pgm.hpp>


#include "cuda/base/thrust.cuh"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The PGM solver namespace.
 *
 * @ingroup pgm
 */
namespace pgm {


#include "common/cuda_hip/multigrid/pgm_kernels.hpp.inc"


}  // namespace pgm
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
