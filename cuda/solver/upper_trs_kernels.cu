// SPDX-FileCopyrightText: 2017-2023 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/solver/upper_trs_kernels.hpp"


#include <memory>


#include <hip/hip_runtime.h>
#include <hipsparse.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/solver/triangular.hpp>


#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/solver/common_trs_kernels.cuh"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The UPPER_TRS solver namespace.
 *
 * @ingroup upper_trs
 */
namespace upper_trs {


void should_perform_transpose(std::shared_ptr<const CudaExecutor> exec,
                              bool& do_transpose)
{
    should_perform_transpose_kernel(exec, do_transpose);
}


template <typename ValueType, typename IndexType>
void generate(std::shared_ptr<const CudaExecutor> exec,
              const matrix::Csr<ValueType, IndexType>* matrix,
              std::shared_ptr<solver::SolveStruct>& solve_struct,
              bool unit_diag, const solver::trisolve_algorithm algorithm,
              const size_type num_rhs)
{
    if (algorithm == solver::trisolve_algorithm::sparselib) {
        generate_kernel<ValueType, IndexType>(exec, matrix, solve_struct,
                                              num_rhs, true, unit_diag);
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_UPPER_TRS_GENERATE_KERNEL);


template <typename ValueType, typename IndexType>
void solve(std::shared_ptr<const CudaExecutor> exec,
           const matrix::Csr<ValueType, IndexType>* matrix,
           const solver::SolveStruct* solve_struct, bool unit_diag,
           const solver::trisolve_algorithm algorithm,
           matrix::Dense<ValueType>* trans_b, matrix::Dense<ValueType>* trans_x,
           const matrix::Dense<ValueType>* b, matrix::Dense<ValueType>* x)
{
    if (algorithm == solver::trisolve_algorithm::sparselib) {
        solve_kernel<ValueType, IndexType>(exec, matrix, solve_struct, trans_b,
                                           trans_x, b, x);
    } else {
        sptrsv_naive_caching<true>(exec, matrix, unit_diag, b, x);
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_UPPER_TRS_SOLVE_KERNEL);


}  // namespace upper_trs
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
