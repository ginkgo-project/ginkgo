#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/batch_bicgstab_kernels.hpp"


#include <ginkgo/core/base/math.hpp>


#include "cuda/base/config.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/matrix/batch_struct.hpp"


namespace gko {
namespace kernels {
namespace cuda {

#define GKO_CUDA_BATCH_USE_DYNAMIC_SHARED_MEM 1
constexpr int default_block_size = 128;
constexpr int sm_multiplier = 4;

/**
 * @brief The batch Bicgstab solver namespace.
 *
 * @ingroup batch_bicgstab
 */
namespace batch_bicgstab {


#include "common/components/uninitialized_array.hpp.inc"
// include all depedencies (note: do not remove this comment)
#include "common/components/reduction.hpp.inc"
#include "common/log/batch_logger.hpp.inc"
#include "common/matrix/batch_csr_kernels.hpp.inc"
#include "common/matrix/batch_vector_kernels.hpp.inc"
#include "common/preconditioner/batch_identity.hpp.inc"
#include "common/preconditioner/batch_jacobi.hpp.inc"
#include "common/solver/batch_bicgstab_kernels.hpp.inc"
#include "common/stop/batch_criteria.hpp.inc"


template <typename T>
using BatchBicgstabOptions =
    gko::kernels::batch_bicgstab::BatchBicgstabOptions<T>;

#define BATCH_BICGSTAB_KERNEL_LAUNCH(_stoppertype, _prectype)                \
    apply_kernel<stop::_stoppertype<ValueType>>                              \
        <<<nbatch, default_block_size, shared_size>>>(                       \
            opts.max_its, opts.residual_tol, logger, _prectype<ValueType>(), \
            nbatch, nrows, shared_gap, nnz, avalues, col_idxs, row_ptrs,     \
            b.stride, b.num_rhs, b.values, x.values)

template <typename BatchMatrixType, typename LogType, typename ValueType>
static void apply_impl(
    std::shared_ptr<const CudaExecutor> exec,
    const BatchBicgstabOptions<remove_complex<ValueType>> opts, LogType logger,
    const BatchMatrixType &a,
    const gko::batch_dense::UniformBatch<const ValueType> &b,
    const gko::batch_dense::UniformBatch<ValueType> &x)
{
    using real_type = gko::remove_complex<ValueType>;
    const size_type nbatch = a.num_batch;
    const int nrows = a.num_rows;
    const int nnz = a.num_nnz;
    const ValueType *const __restrict__ avalues = a.values;
    const int *const __restrict__ col_idxs = a.col_idxs;
    const int *const __restrict__ row_ptrs = a.row_ptrs;
    const int shared_gap = ((a.num_rows - 1) / 32 + 1) * 32;

    int shared_size =
#if GKO_CUDA_BATCH_USE_DYNAMIC_SHARED_MEM
        gko::kernels::batch_bicgstab::local_memory_requirement<ValueType>(
            /*a.num_rows*/ shared_gap, b.num_rhs);
#else
        0;
#endif

    if (opts.preconditioner == gko::preconditioner::batch::type::none) {
#if GKO_CUDA_BATCH_USE_DYNAMIC_SHARED_MEM
        shared_size +=
            BatchIdentity<ValueType>::dynamic_work_size(a.num_rows, a.num_nnz) *
            sizeof(ValueType);
#endif
        if (opts.tol_type == gko::stop::batch::ToleranceType::absolute) {
            BATCH_BICGSTAB_KERNEL_LAUNCH(AbsResidualMaxIter, BatchIdentity);
        } else {
            BATCH_BICGSTAB_KERNEL_LAUNCH(RelResidualMaxIter, BatchIdentity);
        }


    } else if (opts.preconditioner ==
               gko::preconditioner::batch::type::jacobi) {
#if GKO_CUDA_BATCH_USE_DYNAMIC_SHARED_MEM
        shared_size +=
            BatchJacobi<ValueType>::dynamic_work_size(shared_gap, a.num_nnz) *
            sizeof(ValueType);
#endif

        if (opts.tol_type == gko::stop::batch::ToleranceType::absolute) {
            BATCH_BICGSTAB_KERNEL_LAUNCH(AbsResidualMaxIter, BatchJacobi);
        } else {
            BATCH_BICGSTAB_KERNEL_LAUNCH(RelResidualMaxIter, BatchJacobi);
        }


    } else {
        GKO_NOT_IMPLEMENTED;
    }
}


template <typename ValueType>
void apply(std::shared_ptr<const CudaExecutor> exec,
           const BatchBicgstabOptions<remove_complex<ValueType>> &opts,
           const BatchLinOp *const a,
           const matrix::BatchDense<ValueType> *const b,
           matrix::BatchDense<ValueType> *const x,
           log::BatchLogData<ValueType> &logdata)
{
    using cu_value_type = cuda_type<ValueType>;

    // For now, FinalLogger is the only one available
    batch_log::SimpleFinalLogger<remove_complex<ValueType>> logger(
        static_cast<int>(b->get_size().at(0)[1]), opts.max_its,
        logdata.res_norms->get_values(), logdata.iter_counts.get_data());

    const gko::batch_dense::UniformBatch<cu_value_type> x_b =
        get_batch_struct(x);

    if (auto amat = dynamic_cast<const matrix::BatchCsr<ValueType> *>(a)) {
        auto m_b = get_batch_struct(amat);
        auto b_b = get_batch_struct(b);
        apply_impl(exec, opts, logger, m_b, b_b, x_b);
    } else {
        GKO_NOT_SUPPORTED(a);
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_APPLY_KERNEL);


}  // namespace batch_bicgstab
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
