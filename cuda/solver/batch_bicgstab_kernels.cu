#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2017 - 2024 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/solver/batch_bicgstab_kernels.hpp"


#include <thrust/functional.h>
#include <thrust/transform.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>


#include "core/base/batch_struct.hpp"
#include "core/matrix/batch_struct.hpp"
#include "core/solver/batch_dispatch.hpp"
#include "cuda/base/batch_struct.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/kernel_config.hpp"
#include "cuda/base/thrust.cuh"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/components/uninitialized_array.hpp"
#include "cuda/matrix/batch_struct.hpp"


namespace gko {
namespace kernels {
namespace cuda {


// NOTE: this default block size is not used for the main solver kernel.
constexpr int default_block_size = 256;
constexpr int sm_oversubscription = 4;


/**
 * @brief The batch Bicgstab solver namespace.
 *
 * @ingroup batch_bicgstab
 */
namespace batch_bicgstab {


#include "common/cuda_hip/base/batch_multi_vector_kernels.hpp.inc"
#include "common/cuda_hip/components/uninitialized_array.hpp.inc"
#include "common/cuda_hip/matrix/batch_csr_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_dense_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_ell_kernels.hpp.inc"
#include "common/cuda_hip/solver/batch_bicgstab_kernels.hpp.inc"


template <typename StopType, typename PrecType, typename LogType,
          typename BatchMatrixType, typename ValueType>
int get_num_threads_per_block(std::shared_ptr<const DefaultExecutor> exec,
                              const int num_rows)
{
    int num_warps = std::max(num_rows / 4, 2);
    constexpr int warp_sz = static_cast<int>(config::warp_size);
    const int min_block_size = 2 * warp_sz;
    const int device_max_threads =
        ((std::max(num_rows, min_block_size)) / warp_sz) * warp_sz;
    hipFuncAttributes funcattr;
    hipFuncGetAttributes(&funcattr,
                          reinterpret_cast<const void*>(apply_kernel<StopType), 9, true, PrecType, LogType,
                                       BatchMatrixType, ValueType>);
    const int num_regs_used = funcattr.numRegs;
    int max_regs_blk = 0;
    hipDeviceGetAttribute(&max_regs_blk, hipDeviceAttributeMaxRegistersPerBlock,
                           exec->get_device_id());
    const int max_threads_regs =
        ((max_regs_blk / static_cast<int>(num_regs_used)) / warp_sz) * warp_sz;
    int max_threads = std::min(max_threads_regs, device_max_threads);
    max_threads = max_threads <= 1024 ? max_threads : 1024;
    return std::max(std::min(num_warps * warp_sz, max_threads), min_block_size);
}


template <typename StopType, typename PrecType, typename LogType,
          typename BatchMatrixType, typename ValueType>
int get_max_dynamic_shared_memory(std::shared_ptr<const DefaultExecutor> exec)
{
    int shmem_per_sm = 0;
    hipDeviceGetAttribute(&shmem_per_sm,
                           hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
                           exec->get_device_id());
    GKO_ASSERT_NO_CUDA_ERRORS(hipFuncSetAttribute(reinterpret_cast<const void*>(
        apply_kernel<StopType), 9, true, PrecType, LogType, BatchMatrixType,
                     ValueType>,
        hipFuncAttributePreferredSharedMemoryCarveout, 99 /*%*/));
    hipFuncAttributes funcattr;
    hipFuncGetAttributes(&funcattr,
                          reinterpret_cast<const void*>(apply_kernel<StopType), 9, true, PrecType, LogType,
                                       BatchMatrixType, ValueType>);
    return funcattr.maxDynamicSharedSizeBytes;
}


template <typename T>
using settings = gko::kernels::batch_bicgstab::settings<T>;


template <typename CuValueType>
class kernel_caller {
public:
    using value_type = CuValueType;

    kernel_caller(std::shared_ptr<const DefaultExecutor> exec,
                  const settings<remove_complex<value_type>> settings)
        : exec_{std::move(exec)}, settings_{settings}
    {}

    template <typename StopType, const int n_shared,
              const bool prec_shared_bool, typename PrecType, typename LogType,
              typename BatchMatrixType>
    void launch_apply_kernel(
        const gko::kernels::batch_bicgstab::storage_config& sconf,
        LogType& logger, PrecType& prec, const BatchMatrixType& mat,
        const value_type* const __restrict__ b_values,
        value_type* const __restrict__ x_values,
        value_type* const __restrict__ workspace_data, const int& block_size,
        const size_t& shared_size) const
    {
        apply_kernel<StopType, n_shared, prec_shared_bool>
            <<<mat.num_batch_items, block_size, shared_size,
               exec_->get_stream()>>>(sconf, settings_.max_iterations,
                                      settings_.residual_tol, logger, prec, mat,
                                      b_values, x_values, workspace_data);
    }


    template <typename BatchMatrixType, typename PrecType, typename StopType,
              typename LogType>
    void call_kernel(
        LogType logger, const BatchMatrixType& mat, PrecType prec,
        const gko::batch::multi_vector::uniform_batch<const value_type>& b,
        const gko::batch::multi_vector::uniform_batch<value_type>& x) const
    {
        using real_type = gko::remove_complex<value_type>;
        const size_type num_batch_items = mat.num_batch_items;
        constexpr int align_multiple = 8;
        const int padded_num_rows =
            ceildiv(mat.num_rows, align_multiple) * align_multiple;
        auto shem_guard =
            gko::kernels::cuda::detail::shared_memory_config_guard<
                value_type>();
        const int shmem_per_blk =
            get_max_dynamic_shared_memory<StopType, PrecType, LogType,
                                          BatchMatrixType, value_type>(exec_);
        const int block_size =
            get_num_threads_per_block<StopType, PrecType, LogType,
                                      BatchMatrixType, value_type>(
                exec_, mat.num_rows);
        GKO_ASSERT(block_size >= 2 * config::warp_size);

        const size_t prec_size =
            PrecType::dynamic_work_size(padded_num_rows,
                                        mat.get_single_item_num_nnz()) *
            sizeof(value_type);
        const auto sconf =
            gko::kernels::batch_bicgstab::compute_shared_storage<PrecType,
                                                                 value_type>(
                shmem_per_blk, padded_num_rows, mat.get_single_item_num_nnz(),
                b.num_rhs);
        const size_t shared_size =
            sconf.n_shared * padded_num_rows * sizeof(value_type) +
            (sconf.prec_shared ? prec_size : 0);
        auto workspace = gko::array<value_type>(
            exec_,
            sconf.gmem_stride_bytes * num_batch_items / sizeof(value_type));
        assert(sconf.gmem_stride_bytes % sizeof(value_type) == 0);

        value_type* const workspace_data = workspace.get_data();

        // Template parameters launch_apply_kernel<StopType, n_shared,
        // prec_shared>
        if (sconf.prec_shared) {
            launch_apply_kernel<StopType, 9, true>(
                sconf, logger, prec, mat, b.values, x.values, workspace_data,
                block_size, shared_size);
        } else {
            switch (sconf.n_shared) {
            case 0:
                launch_apply_kernel<StopType, 0, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 1:
                launch_apply_kernel<StopType, 1, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 2:
                launch_apply_kernel<StopType, 2, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 3:
                launch_apply_kernel<StopType, 3, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 4:
                launch_apply_kernel<StopType, 4, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 5:
                launch_apply_kernel<StopType, 5, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 6:
                launch_apply_kernel<StopType, 6, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 7:
                launch_apply_kernel<StopType, 7, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 8:
                launch_apply_kernel<StopType, 8, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            case 9:
                launch_apply_kernel<StopType, 9, false>(
                    sconf, logger, prec, mat, b.values, x.values,
                    workspace_data, block_size, shared_size);
                break;
            default:
                GKO_NOT_IMPLEMENTED;
            }
        }
    }

private:
    std::shared_ptr<const DefaultExecutor> exec_;
    const settings<remove_complex<value_type>> settings_;
};


template <typename ValueType>
void apply(std::shared_ptr<const DefaultExecutor> exec,
           const settings<remove_complex<ValueType>>& settings,
           const batch::BatchLinOp* const mat,
           const batch::BatchLinOp* const precon,
           const batch::MultiVector<ValueType>* const b,
           batch::MultiVector<ValueType>* const x,
           batch::log::detail::log_data<remove_complex<ValueType>>& logdata)
{
    using cu_value_type = cuda_type<ValueType>;
    auto dispatcher = batch::solver::create_dispatcher<ValueType>(
        kernel_caller<cu_value_type>(exec, settings), settings, mat, precon);
    dispatcher.apply(b, x, logdata);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_BICGSTAB_APPLY_KERNEL);


}  // namespace batch_bicgstab
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
