#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2018

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/bicgstab_kernels.hpp"


#include "core/base/exception_helpers.hpp"
#include "core/base/math.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
namespace bicgstab {


constexpr int default_block_size = 512;


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void initialize_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    const ValueType *__restrict__ b, ValueType *__restrict__ r,
    ValueType *__restrict__ rr, ValueType *__restrict__ y,
    ValueType *__restrict__ s, ValueType *__restrict__ t,
    ValueType *__restrict__ z, ValueType *__restrict__ v,
    ValueType *__restrict__ p, ValueType *__restrict__ prev_rho,
    ValueType *__restrict__ rho, ValueType *__restrict__ alpha,
    ValueType *__restrict__ beta, ValueType *__restrict__ gamma,
    ValueType *__restrict__ omega, stopping_status *__restrict__ stop_status)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;

    if (tidx < num_cols) {
        prev_rho[tidx] = one<ValueType>();
        rho[tidx] = one<ValueType>();
        alpha[tidx] = one<ValueType>();
        beta[tidx] = one<ValueType>();
        gamma[tidx] = one<ValueType>();
        omega[tidx] = one<ValueType>();
        stop_status[tidx].reset();
    }

    if (tidx < num_rows * stride) {
        r[tidx] = b[tidx];
        rr[tidx] = zero<ValueType>();
        y[tidx] = zero<ValueType>();
        s[tidx] = zero<ValueType>();
        t[tidx] = zero<ValueType>();
        z[tidx] = zero<ValueType>();
        v[tidx] = zero<ValueType>();
        p[tidx] = zero<ValueType>();
    }
}


template <typename ValueType>
void initialize(std::shared_ptr<const CudaExecutor> exec,
                const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *r,
                matrix::Dense<ValueType> *rr, matrix::Dense<ValueType> *y,
                matrix::Dense<ValueType> *s, matrix::Dense<ValueType> *t,
                matrix::Dense<ValueType> *z, matrix::Dense<ValueType> *v,
                matrix::Dense<ValueType> *p, matrix::Dense<ValueType> *prev_rho,
                matrix::Dense<ValueType> *rho, matrix::Dense<ValueType> *alpha,
                matrix::Dense<ValueType> *beta, matrix::Dense<ValueType> *gamma,
                matrix::Dense<ValueType> *omega,
                Array<stopping_status> *stop_status)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(b->get_size()[0] * b->get_stride(), block_size.x), 1, 1);

    initialize_kernel<<<grid_size, block_size, 0, 0>>>(
        b->get_size()[0], b->get_size()[1], b->get_stride(),
        as_cuda_type(b->get_const_values()), as_cuda_type(r->get_values()),
        as_cuda_type(rr->get_values()), as_cuda_type(y->get_values()),
        as_cuda_type(s->get_values()), as_cuda_type(t->get_values()),
        as_cuda_type(z->get_values()), as_cuda_type(v->get_values()),
        as_cuda_type(p->get_values()), as_cuda_type(prev_rho->get_values()),
        as_cuda_type(rho->get_values()), as_cuda_type(alpha->get_values()),
        as_cuda_type(beta->get_values()), as_cuda_type(gamma->get_values()),
        as_cuda_type(omega->get_values()),
        as_cuda_type(stop_status->get_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_INITIALIZE_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void step_1_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    const ValueType *__restrict__ r, ValueType *__restrict__ p,
    const ValueType *__restrict__ v, const ValueType *__restrict__ rho,
    const ValueType *__restrict__ prev_rho, const ValueType *__restrict__ alpha,
    const ValueType *__restrict__ omega,
    const stopping_status *__restrict__ stop_status)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto col = tidx % stride;
    if (col >= num_cols || tidx >= num_rows * stride ||
        stop_status[col].has_stopped()) {
        return;
    }
    auto res = r[tidx];
    if (prev_rho[col] * omega[col] != zero<ValueType>()) {
        const auto tmp = (rho[col] / prev_rho[col]) * (alpha[col] / omega[col]);
        res += tmp * (p[tidx] - omega[col] * v[tidx]);
    }
    p[tidx] = res;
}


template <typename ValueType>
void step_1(std::shared_ptr<const CudaExecutor> exec,
            const matrix::Dense<ValueType> *r, matrix::Dense<ValueType> *p,
            const matrix::Dense<ValueType> *v,
            const matrix::Dense<ValueType> *rho,
            const matrix::Dense<ValueType> *prev_rho,
            const matrix::Dense<ValueType> *alpha,
            const matrix::Dense<ValueType> *omega,
            const Array<stopping_status> *stop_status)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(r->get_size()[0] * r->get_stride(), block_size.x), 1, 1);

    step_1_kernel<<<grid_size, block_size, 0, 0>>>(
        r->get_size()[0], r->get_size()[1], r->get_stride(),
        as_cuda_type(r->get_const_values()), as_cuda_type(p->get_values()),
        as_cuda_type(v->get_const_values()),
        as_cuda_type(rho->get_const_values()),
        as_cuda_type(prev_rho->get_const_values()),
        as_cuda_type(alpha->get_const_values()),
        as_cuda_type(omega->get_const_values()),
        as_cuda_type(stop_status->get_const_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_STEP_1_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void step_2_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    const ValueType *__restrict__ r, ValueType *__restrict__ s,
    const ValueType *__restrict__ v, const ValueType *__restrict__ rho,
    ValueType *__restrict__ alpha, const ValueType *__restrict__ beta,
    const stopping_status *__restrict__ stop_status)
{
    const size_type tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const size_type col = tidx % stride;
    if (col >= num_cols || tidx >= num_rows * stride ||
        stop_status[col].has_stopped()) {
        return;
    }
    auto t_alpha = zero<ValueType>();
    auto t_s = r[tidx];
    if (beta[col] != zero<ValueType>()) {
        t_alpha = rho[col] / beta[col];
        t_s -= t_alpha * v[tidx];
    }
    alpha[col] = t_alpha;
    s[tidx] = t_s;
}


template <typename ValueType>
void step_2(std::shared_ptr<const CudaExecutor> exec,
            const matrix::Dense<ValueType> *r, matrix::Dense<ValueType> *s,
            const matrix::Dense<ValueType> *v,
            const matrix::Dense<ValueType> *rho,
            matrix::Dense<ValueType> *alpha,
            const matrix::Dense<ValueType> *beta,
            const Array<stopping_status> *stop_status)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(r->get_size()[0] * r->get_stride(), block_size.x), 1, 1);

    step_2_kernel<<<grid_size, block_size, 0, 0>>>(
        r->get_size()[0], r->get_size()[1], r->get_stride(),
        as_cuda_type(r->get_const_values()), as_cuda_type(s->get_values()),
        as_cuda_type(v->get_const_values()),
        as_cuda_type(rho->get_const_values()),
        as_cuda_type(alpha->get_values()),
        as_cuda_type(beta->get_const_values()),
        as_cuda_type(stop_status->get_const_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_STEP_2_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void step_3_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    size_type x_stride, ValueType *__restrict__ x, ValueType *__restrict__ r,
    const ValueType *__restrict__ s, const ValueType *__restrict__ t,
    const ValueType *__restrict__ y, const ValueType *__restrict__ z,
    const ValueType *__restrict__ alpha, const ValueType *__restrict__ beta,
    const ValueType *__restrict__ gamma, ValueType *__restrict__ omega,
    const stopping_status *__restrict__ stop_status)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto row = tidx / stride;
    const auto col = tidx % stride;
    if (col >= num_cols || tidx >= num_rows * stride ||
        stop_status[col].has_stopped()) {
        return;
    }
    const auto x_pos = row * x_stride + col;
    auto t_omega = zero<ValueType>();
    auto t_x = x[x_pos] + alpha[col] * y[tidx];
    auto t_r = s[tidx];
    if (beta[col] != zero<ValueType>()) {
        t_omega = gamma[col] / beta[col];
        t_x += t_omega * z[tidx];
        t_r -= t_omega * t[tidx];
    }
    omega[col] = t_omega;
    x[x_pos] = t_x;
    r[tidx] = t_r;
}


template <typename ValueType>
void step_3(
    std::shared_ptr<const CudaExecutor> exec, matrix::Dense<ValueType> *x,
    matrix::Dense<ValueType> *r, const matrix::Dense<ValueType> *s,
    const matrix::Dense<ValueType> *t, const matrix::Dense<ValueType> *y,
    const matrix::Dense<ValueType> *z, const matrix::Dense<ValueType> *alpha,
    const matrix::Dense<ValueType> *beta, const matrix::Dense<ValueType> *gamma,
    matrix::Dense<ValueType> *omega, const Array<stopping_status> *stop_status)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(r->get_size()[0] * r->get_stride(), block_size.x), 1, 1);

    step_3_kernel<<<grid_size, block_size, 0, 0>>>(
        r->get_size()[0], r->get_size()[1], r->get_stride(), x->get_stride(),
        as_cuda_type(x->get_values()), as_cuda_type(r->get_values()),
        as_cuda_type(s->get_const_values()),
        as_cuda_type(t->get_const_values()),
        as_cuda_type(y->get_const_values()),
        as_cuda_type(z->get_const_values()),
        as_cuda_type(alpha->get_const_values()),
        as_cuda_type(beta->get_const_values()),
        as_cuda_type(gamma->get_const_values()),
        as_cuda_type(omega->get_values()),
        as_cuda_type(stop_status->get_const_data()));
}
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_STEP_3_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void finalize_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    size_type x_stride, ValueType *__restrict__ x,
    const ValueType *__restrict__ y, const ValueType *__restrict__ alpha,
    stopping_status *__restrict__ stop_status)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto row = tidx / stride;
    const auto col = tidx % stride;
    if (col >= num_cols || tidx >= num_rows * stride ||
        stop_status[col].is_finalized() || !stop_status[col].has_stopped()) {
        return;
    }
    const auto x_pos = row * x_stride + col;
    x[x_pos] = x[x_pos] + alpha[col] * y[tidx];
    stop_status[col].finalize();
}


template <typename ValueType>
void finalize(std::shared_ptr<const CudaExecutor> exec,
              matrix::Dense<ValueType> *x, const matrix::Dense<ValueType> *y,
              const matrix::Dense<ValueType> *alpha,
              Array<stopping_status> *stop_status)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(y->get_size()[0] * y->get_stride(), block_size.x), 1, 1);

    finalize_kernel<<<grid_size, block_size, 0, 0>>>(
        y->get_size()[0], y->get_size()[1], y->get_stride(), x->get_stride(),
        as_cuda_type(x->get_values()), as_cuda_type(y->get_const_values()),
        as_cuda_type(alpha->get_const_values()),
        as_cuda_type(stop_status->get_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_FINALIZE_KERNEL);


}  // namespace bicgstab
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
