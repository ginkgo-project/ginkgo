#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2023, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/batch_gmres_kernels.hpp"


#include <ginkgo/batch_config.hpp>
#include <ginkgo/core/base/math.hpp>


#include "core/solver/batch_dispatch.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/exception.cuh"
#include "cuda/base/kernel_config.cuh"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/matrix/batch_struct.hpp"


namespace gko {
namespace kernels {
namespace cuda {

constexpr int default_block_size = 128;
constexpr int sm_multiplier = 4;

/**
 * @brief The batch Gmres solver namespace.
 *
 * @ingroup batch_gmres
 */
namespace batch_gmres {


#include "common/cuda_hip/components/uninitialized_array.hpp.inc"
// include all depedencies (note: do not remove this comment)
#include "common/cuda_hip/matrix/batch_csr_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_dense_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_ell_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_vector_kernels.hpp.inc"
#include "common/cuda_hip/solver/batch_gmres_kernels.hpp.inc"

int get_larger_power(int value, int guess = 64)
{
    return guess >= value ? guess : get_larger_power(value, guess << 1);
}

template <typename StopType, typename PrecType, typename LogType,
          typename BatchMatrixType, typename ValueType>
int get_num_threads_per_block(std::shared_ptr<const CudaExecutor> exec,
                              const int num_rows)
{
    int nwarps = num_rows / 4;
    if (nwarps < 2) {
        nwarps = 2;
    }
    const int min_block_size = 2 * config::warp_size;
    const int device_max_threads =
        ((std::max(num_rows, min_block_size)) / config::warp_size) *
        config::warp_size;
    hipFuncAttributes funcattr;
    hipFuncGetAttributes(&funcattr,
                          reinterpret_cast<const void*>(apply_kernel<StopType), 0, 0, PrecType, LogType,
                                       BatchMatrixType, ValueType>);
    const int num_regs_used = funcattr.numRegs;
    int max_regs_blk = 0;
    hipDeviceGetAttribute(&max_regs_blk, hipDeviceAttributeMaxRegistersPerBlock,
                           exec->get_device_id());
    // FIXME: Using magic number, 1.1
    const int max_threads_regs =
        ((max_regs_blk /
          static_cast<int>((static_cast<double>(num_regs_used) * 1.1))) /
         config::warp_size) *
        config::warp_size;
    int max_threads = std::min(max_threads_regs, device_max_threads);
    max_threads = max_threads <= 1024 ? max_threads : 1024;
    return std::min(nwarps * static_cast<int>(config::warp_size), max_threads);
}


template <typename StopType, typename PrecType, typename LogType,
          typename BatchMatrixType, typename ValueType>
int get_max_dynamic_shared_memory(std::shared_ptr<const CudaExecutor> exec,
                                  const size_type required_cache_storage)
{
    int shmem_per_sm = 0;
    hipDeviceGetAttribute(&shmem_per_sm,
                           hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
                           exec->get_device_id());
    // std::cerr << " Max shared mem per SM = " << shmem_per_sm << std::endl;
    // int max_shared_pc =
    //     100 - static_cast<int>(static_cast<double>(required_cache_storage) /
    //                            shmem_per_sm * 100);
    // if (max_shared_pc <= 0) {
    //     max_shared_pc = 1;
    // }
    // // std::cerr << " Max shared pc required = " << max_shared_pc <<
    // std::endl; GKO_ASSERT_NO_CUDA_ERRORS(hipFuncSetAttribute(reinterpret_cast<const void*>(
    //     apply_kernel<StopType), 11, 1, PrecType, LogType, BatchMatrixType,
    //     ValueType>, hipFuncAttributePreferredSharedMemoryCarveout,
    //     max_shared_pc - 1));
    // hipFuncAttributes funcattr;
    // hipFuncGetAttributes(
    //     &funcattr,
    reinterpret_cast<const void*>(//     apply_kernel<StopType), 11, 1, PrecType, LogType, BatchMatrixType,
    //     ValueType>);
    // std::cerr << " Max dyn. shared memory for batch bcgs = ",
    //        << funcattr.maxDynamicSharedSizeBytes << std::endl;
    // return funcattr.maxDynamicSharedSizeBytes;
    return shmem_per_sm;
}


template <typename T>
using BatchGmresOptions = gko::kernels::batch_gmres::BatchGmresOptions<T>;


template <typename CuValueType>
class KernelCaller {
public:
    using value_type = CuValueType;

    KernelCaller(std::shared_ptr<const CudaExecutor> exec,
                 const BatchGmresOptions<remove_complex<value_type>> opts)
        : exec_{exec}, opts_{opts}
    {}

    template <typename StopType, const int n_shared,
              const bool prec_shared_bool, typename PrecType, typename LogType,
              typename BatchMatrixType>
    void launch_apply_kernel(
        const gko::kernels::batch_gmres::StorageConfig& sconf, LogType& logger,
        PrecType& prec, const BatchMatrixType& a,
        const value_type* const __restrict__ b_values,
        value_type* const __restrict__ x_values,
        value_type* const __restrict__ workspace_data, const int& block_size,
        const size_t& shared_size) const
    {
        auto nrows = a.num_rows;

        apply_kernel<StopType, n_shared, prec_shared_bool>
            <<<a.num_batch, block_size, shared_size>>>(
                sconf, opts_.max_its, opts_.residual_tol, opts_.restart_num,
                logger, prec, a, b_values, x_values, workspace_data);
    }

    template <typename BatchMatrixType, typename PrecType, typename StopType,
              typename LogType>
    void call_kernel(LogType logger, const BatchMatrixType& a, PrecType prec,
                     const gko::batch_dense::UniformBatch<const value_type>& b,
                     const gko::batch_dense::UniformBatch<value_type>& x) const
    {
        using real_type = gko::remove_complex<value_type>;
        const size_type nbatch = a.num_batch;
        const auto restart = opts_.restart_num;
        constexpr int align_multiple = 8;
        const int shared_gap =
            ((a.num_rows - 1) / align_multiple + 1) * align_multiple;
        gko::kernels::cuda::configure_shared_memory_banks<value_type>();

        const int shmem_per_blk = 0;
        // get_max_dynamic_shared_memory<StopType, PrecType, LogType,
        //                               BatchMatrixType, value_type>(exec_,
        //                                                            0);

        const int block_size = 128;
        // get_num_threads_per_block<StopType, PrecType, LogType,
        //                           BatchMatrixType, value_type>(exec_,
        //                                                        a.num_rows);
        assert(block_size >= 2 * config::warp_size);

        const size_t prec_size =
            PrecType::dynamic_work_size(shared_gap, a.num_nnz);
        const size_t subspace_size = a.num_rows * (restart + 1);
        const size_t hess_size = restart * (restart + 2);
        const auto sconf =
            gko::kernels::batch_gmres::compute_shared_storage<PrecType,
                                                              value_type>(
                shmem_per_blk, shared_gap, a.num_nnz, b.num_rhs, restart);
        int num_main_vecs_shared = min(sconf.n_shared, 5);
        int num_rot_vecs_shared = min(sconf.n_shared - num_main_vecs_shared, 4);

        std::cout << "HERE  " << num_main_vecs_shared << " "
                  << num_rot_vecs_shared << std::endl;
        std::cout << "HERE  " << sconf.hess_shared << " "
                  << sconf.subspace_shared << " " << sconf.prec_shared
                  << std::endl;
        std::cout << sconf.gmem_stride_bytes << std::endl;

        const size_t shared_size =
            (num_main_vecs_shared * shared_gap +
             num_rot_vecs_shared * (restart + 1) +
             (sconf.prec_shared ? prec_size : 0) +
             (sconf.subspace_shared ? subspace_size : 0) +
             (sconf.hess_shared ? hess_size : 0)) *
            sizeof(value_type);
        auto workspace = gko::array<value_type>(
            exec_, sconf.gmem_stride_bytes * nbatch / sizeof(value_type));
        assert(sconf.gmem_stride_bytes % sizeof(value_type) == 0);

        value_type* const workspace_data = workspace.get_data();
        int n_shared = sconf.n_shared + int(sconf.hess_shared) +
                       int(sconf.subspace_shared);
        auto prec_shared_bool = sconf.prec_shared;

        // Template for calling launch_apply_kernel:
        // < StopType, n_shared, prec_shared_bool>
        if (prec_shared_bool) {
            switch (n_shared) {
            case 11:
                launch_apply_kernel<StopType, 11, 1>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 10:
                launch_apply_kernel<StopType, 10, 1>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 9:
                launch_apply_kernel<StopType, 9, 1>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            }
        } else {
            switch (n_shared) {
            case 0:
                launch_apply_kernel<StopType, 0, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 1:
                launch_apply_kernel<StopType, 1, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 2:
                launch_apply_kernel<StopType, 2, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 3:
                launch_apply_kernel<StopType, 3, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 4:
                launch_apply_kernel<StopType, 4, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 5:
                launch_apply_kernel<StopType, 5, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 6:
                launch_apply_kernel<StopType, 6, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 7:
                launch_apply_kernel<StopType, 7, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 8:
                launch_apply_kernel<StopType, 8, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 9:
                launch_apply_kernel<StopType, 9, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 10:
                launch_apply_kernel<StopType, 10, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            case 11:
                launch_apply_kernel<StopType, 11, 0>(
                    sconf, logger, prec, a, b.values, x.values, workspace_data,
                    block_size, shared_size);
                break;
            }
        }
        GKO_CUDA_LAST_IF_ERROR_THROW;
    }

private:
    std::shared_ptr<const CudaExecutor> exec_;
    const BatchGmresOptions<remove_complex<value_type>> opts_;
};


template <typename ValueType>
void apply(std::shared_ptr<const CudaExecutor> exec,
           const BatchGmresOptions<remove_complex<ValueType>>& opts,
           const BatchLinOp* const a, const BatchLinOp* const precon,
           const matrix::BatchDense<ValueType>* const b,
           matrix::BatchDense<ValueType>* const x,
           log::BatchLogData<ValueType>& logdata)
{
    using cu_value_type = cuda_type<ValueType>;
    auto dispatcher = batch_solver::create_dispatcher<ValueType>(
        KernelCaller<cu_value_type>(exec, opts), opts, a, precon);
    dispatcher.apply(b, x, logdata);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_GMRES_APPLY_KERNEL);


}  // namespace batch_gmres
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
