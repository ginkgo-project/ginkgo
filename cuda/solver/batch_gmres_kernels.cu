#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/batch_gmres_kernels.hpp"


#include <ginkgo/batch_config.hpp>
#include <ginkgo/core/base/math.hpp>


#include "core/solver/batch_dispatch.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/exception.cuh"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/matrix/batch_struct.hpp"


namespace gko {
namespace kernels {
namespace cuda {


constexpr int default_block_size = 128;
constexpr int sm_multiplier = 4;

/**
 * @brief The batch Gmres solver namespace.
 *
 * @ingroup batch_gmres
 */
namespace batch_gmres {


#include "common/cuda_hip/components/uninitialized_array.hpp.inc"
// include all depedencies (note: do not remove this comment)
#include "common/cuda_hip/matrix/batch_csr_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_dense_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_ell_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_vector_kernels.hpp.inc"
#include "common/cuda_hip/solver/batch_gmres_kernels.hpp.inc"


template <typename T>
using BatchGmresOptions = gko::kernels::batch_gmres::BatchGmresOptions<T>;


template <typename CuValueType>
class KernelCaller {
public:
    using value_type = CuValueType;

    KernelCaller(std::shared_ptr<const CudaExecutor> exec,
                 const BatchGmresOptions<remove_complex<value_type>> opts)
        : exec_{exec}, opts_{opts}
    {}

    template <typename BatchMatrixType, typename PrecType, typename StopType,
              typename LogType>
    void call_kernel(LogType logger, const BatchMatrixType& a,
                     const gko::batch_dense::UniformBatch<const value_type>& b,
                     const gko::batch_dense::UniformBatch<value_type>& x) const
    {
        using real_type = gko::remove_complex<value_type>;
        const size_type nbatch = a.num_batch;
        const value_type* const bptr = b.values;
        value_type* const xptr = x.values;

        static_assert(default_block_size >= 2 * config::warp_size,
                      "Need at least two warps per block!");

        const auto nrhs = b.num_rhs;
        const auto nrows = a.num_rows;
        const auto restart = opts_.restart_num;
        const int global_gap =
            6 * nrows * nrhs + 3 * restart * nrhs + (restart + 1) * nrhs +
            restart * (restart + 1) * nrhs + nrows * (restart + 1) * nrhs;
        auto workspace = gko::Array<value_type>(exec_);

        const int shared_size =
            gko::kernels::batch_gmres::local_memory_requirement<value_type>(
                a.num_rows, b.num_rhs, opts_.restart_num) +
            PrecType::dynamic_work_size(a.num_rows, a.num_nnz) *
                sizeof(value_type);
#if GKO_CUDA_BATCH_GMRES_HAVE_NO_SHMEM
        workspace = gko::Array<value_type>(
            exec_,
            static_cast<size_type>(shared_size * nbatch / sizeof(value_type)));
        apply_kernel<StopType><<<nbatch, default_block_size>>>(
            global_gap, opts_.max_its, opts_.residual_tol, opts_.restart_num,
            logger, PrecType(), a, bptr, xptr, workspace.get_data());
#else
        apply_kernel<StopType><<<nbatch, default_block_size, shared_size>>>(
            global_gap, opts_.max_its, opts_.residual_tol, opts_.restart_num,
            logger, PrecType(), a, bptr, xptr);
#endif
        GKO_CUDA_LAST_IF_ERROR_THROW;
    }

private:
    std::shared_ptr<const CudaExecutor> exec_;
    const BatchGmresOptions<remove_complex<value_type>> opts_;
};


template <typename ValueType>
void apply(std::shared_ptr<const CudaExecutor> exec,
           const BatchGmresOptions<remove_complex<ValueType>>& opts,
           const BatchLinOp* const a,
           const matrix::BatchDense<ValueType>* const b,
           matrix::BatchDense<ValueType>* const x,
           log::BatchLogData<ValueType>& logdata)
{
    using cu_value_type = cuda_type<ValueType>;
    auto dispatcher = batch_solver::create_dispatcher<ValueType>(
        KernelCaller<cu_value_type>(exec, opts), opts);
    dispatcher.apply(a, b, x, logdata);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_GMRES_APPLY_KERNEL);


}  // namespace batch_gmres
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
