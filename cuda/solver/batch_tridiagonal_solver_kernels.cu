#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2023, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/batch_tridiagonal_solver_kernels.hpp"


#include "core/matrix/batch_struct.hpp"
#include "cuda/base/cublas_bindings.hpp"
#include "cuda/base/exception.cuh"
#include "cuda/components/load_store.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/matrix/batch_struct.hpp"

namespace gko {
namespace kernels {
namespace cuda {
namespace batch_tridiagonal_solver {

namespace {

constexpr int default_subwarp_size = config::warp_size;
constexpr int default_block_size =
    128;  // found out by experimentally that 128 works the best

}  // namespace

namespace {

template <typename ValueType, typename Group>
__device__ void broadcast(Group& subwarp_grp, const int target_lane,
                          const ValueType& my_a, const ValueType& my_b,
                          const ValueType& my_c, const ValueType& my_d,
                          ValueType& piv_a, ValueType& piv_b, ValueType& piv_c,
                          ValueType& piv_d)
{
    piv_a = subwarp_grp.shfl(my_a, target_lane);
    piv_b = subwarp_grp.shfl(my_b, target_lane);
    piv_c = subwarp_grp.shfl(my_c, target_lane);
    piv_d = subwarp_grp.shfl(my_d, target_lane);
}

template <typename ValueType, typename Group>
__device__ void WM_step(Group& subwarp_grp, const int curr_group_size,
                        const int tile_size, ValueType& my_a, ValueType& my_b,
                        ValueType& my_c, ValueType& my_d)
{
    const int lane = subwarp_grp.thread_rank();
    const int curr_grp_idx = lane / curr_group_size;

    const bool is_left_grp = (curr_grp_idx % 2) == 0;
    ValueType piv_a, piv_b, piv_c, piv_d;
    ValueType my_f = zero<ValueType>();

    const int left_grp_last_lane =
        is_left_grp ? ((curr_grp_idx + 1) * curr_group_size) - 1
                    : (curr_grp_idx * curr_group_size) - 1;
    const int right_grp_first_lane = left_grp_last_lane + 1;

    // broadcast last equation of the left group
    broadcast(subwarp_grp, left_grp_last_lane, my_a, my_b, my_c, my_d, piv_a,
              piv_b, piv_c, piv_d);

    if (lane == right_grp_first_lane) {
        my_f = my_b;
    }

    // eliminate a of the right group
    if (!is_left_grp) {
        const ValueType mult = my_a / piv_b;
        my_a = -1 * piv_a * mult;
        my_d -= piv_d * mult;
        my_f -= piv_c * mult;
    }
    if (lane == right_grp_first_lane) {
        my_b = my_f;
    }

    // broadcast first equation of the right group
    broadcast(subwarp_grp, right_grp_first_lane, my_a, my_b, my_c, my_d, piv_a,
              piv_b, piv_c, piv_d);

    // eliminate c of the left group
    if (is_left_grp) {
        const ValueType mult = my_c / piv_b;
        my_a -= piv_a * mult;
        my_d -= piv_d * mult;
        my_c = -1 * piv_c * mult;
    }

    // eliminate fill-in of the right group except for its first row
    if (!is_left_grp && lane != right_grp_first_lane) {
        const ValueType mult = my_f / piv_b;
        my_a -= piv_a * mult;
        my_d -= piv_d * mult;
        my_c -= piv_c * mult;
    }
}

template <typename ValueType, typename Group>
__device__ void WM_phase(const int num_WM_steps, Group& subwarp_grp,
                         const int tile_size, ValueType& my_a, ValueType& my_b,
                         ValueType& my_c, ValueType& my_d, int& curr_group_size)
{
    for (int i = 0; i < num_WM_steps; i++) {
        WM_step(curr_group_size, tile_size, my_a, my_b, my_c, my_d);
        curr_group_size *= 2;
        subwarp_grp.sync();
    }
}

template <int subwarp_size, typename ValueType>
__global__ void WM_pGE_kernel_approach_1(const int num_WM_steps,
                                         const size_type nbatch,
                                         const int nrows, ValueType* const a,
                                         ValueType* const b, ValueType* const c,
                                         ValueType* const d, ValueType* const x)
{
    auto subwarpgrp =
        group::tiled_partition<subwarp_size>(group::this_thread_block());
    const int subgrpwarp_id_in_grid =
        thread::get_subwarp_id_flat<subwarp_size, int>();
    const int total_num_subwarp_grps_in_grid =
        thread::get_subwarp_num_flat<subwarp_size, int>();
    const int id_within_warp = subwarpgrp.thread_rank();

    // a subwarp per matrix in the batch
    for (size_type batch_idx = subgrpwarp_id_in_grid; batch_idx < nbatch;
         batch_idx += total_num_subwarp_grps_in_grid) {
        // Approach: a thread in the subwarp handles one row of the matrix or to
        // be precise, a row in the matrix tile
        const auto tile_size = subwarp_size;
        const auto num_tiles = ceildiv(nrows, tile_size);
        const bool is_last_tile_similar = ((nrows % tile_size) == 0);
        assert(pow(2, num_WM_steps) <= tile_size);

        for (int tile_id = 0; tile_id < num_tiles; tile_id++) {
            const int row_idx_st_tile = tile_id * tile_size;  // inclusive
            const int row_idx_end_tile =
                tile_id == num_tiles - 1
                    ? nrows
                    : (tile_id + 1) * tile_size;  // exclusive

            ValueType my_a;
            ValueType my_b;
            ValueType my_c;
            ValueType my_d;

            const int my_row_idx = row_idx_st_tile + id_within_warp;

            if (my_row_idx < row_idx_end_tile) {
                my_a = a[batch_idx * nrows + my_row_idx];
                my_b = b[batch_idx * nrows + my_row_idx];
                my_c = c[batch_idx * nrows + my_row_idx];
                my_d = d[batch_idx * nrows + my_row_idx];
                // coalesced accesses while reading data
            }

            // TODO: last tile - if non-similar ???

            // Phase-1 of the alogithm- WM phase
            int curr_group_size = 1;
            WM_phase(num_WM_steps, subwarpgrp, tile_size, my_a, my_b, my_c,
                     my_d, curr_group_size);
            // In each WM step, the adjacent groups are merged independently.

            // Phase-2 of the algorithm - Full Gaussean elimination of the
            // groups Now perform full Gaussean elimination on each group of the
            // transformed system to eliminate the bottom spikes
            // Forward_full_GE_phase();
        }
    }
}

}  // namespace


template <typename ValueType>
void apply(std::shared_ptr<const DefaultExecutor> exec,
           matrix::BatchTridiagonal<ValueType>* const tridiag_mat,
           matrix::BatchDense<ValueType>* const rhs,
           matrix::BatchDense<ValueType>* const x)
{
    const auto nbatch = tridiag_mat->get_num_batch_entries();
    const auto nrows = static_cast<int>(tridiag_mat->get_size().at(0)[0]);
    const auto nrhs = rhs->get_size().at(0)[1];
    assert(nrhs == 1);

    const int shared_size =
        gko::kernels::batch_tridiagonal_solver::local_memory_requirement<
            ValueType>(nrows, nrhs);

    const auto subwarpsize = default_subwarp_size;
    dim3 block(default_block_size);
    dim3 grid(ceildiv(nbatch * subwarpsize, default_block_size));

    const int num_WM_steps = 2;

    WM_pGE_kernel_approach_1<subwarpsize><<<grid, block, shared_size>>>(
        num_WM_steps, nbatch, nrows, tridiag_mat->get_sub_diagonal(),
        tridiag_mat->get_main_diagonal(), tridiag_mat->get_super_diagonal(),
        rhs->get_values(), x->get_values());

    GKO_CUDA_LAST_IF_ERROR_THROW;
}


GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_BATCH_TRIDIAGONAL_SOLVER_APPLY_KERNEL);


}  // namespace batch_tridiagonal_solver
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
