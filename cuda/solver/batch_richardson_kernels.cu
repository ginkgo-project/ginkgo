#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2022, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/batch_richardson_kernels.hpp"


#include <ginkgo/core/base/math.hpp>


#include "core/solver/batch_dispatch.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/exception.cuh"
#include "cuda/base/kernel_config.cuh"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/intrinsics.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/matrix/batch_struct.hpp"


namespace gko {
namespace kernels {
namespace cuda {


#define GKO_CUDA_BATCH_USE_DYNAMIC_SHARED_MEM 1
constexpr int default_block_size = 256;
constexpr int sm_multiplier = 4;


/**
 * @brief The batch Richardson solver namespace.
 *
 * @ingroup batch_rich
 */
namespace batch_rich {


#include "common/cuda_hip/components/uninitialized_array.hpp.inc"
// include all depedencies (note: do not remove this comment)
#include "common/cuda_hip/matrix/batch_csr_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_dense_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_ell_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_vector_kernels.hpp.inc"
#include "common/cuda_hip/solver/batch_richardson_kernels.hpp.inc"


template <typename T>
using BatchRichardsonOptions =
    gko::kernels::batch_rich::BatchRichardsonOptions<T>;


template <typename CuValueType>
class KernelCaller {
public:
    using value_type = CuValueType;

    KernelCaller(std::shared_ptr<const CudaExecutor> exec,
                 const BatchRichardsonOptions<remove_complex<value_type>> opts)
        : exec_{exec}, opts_{opts}
    {}

    template <typename BatchMatrixType, typename PrecType, typename StopType,
              typename LogType>
    void call_kernel(LogType logger, const BatchMatrixType& a,
                     const gko::batch_dense::UniformBatch<const value_type>& b,
                     const gko::batch_dense::UniformBatch<value_type>& x) const
    {
        using real_type = gko::remove_complex<value_type>;
        const size_type nbatch = a.num_batch;

        const int shared_size =
            gko::kernels::batch_rich::local_memory_requirement<value_type>(
                a.num_rows, b.num_rhs) +
            PrecType::dynamic_work_size(a.num_rows, a.num_nnz) *
                sizeof(value_type);

        apply_kernel<StopType><<<nbatch, default_block_size, shared_size>>>(
            opts_.max_its, opts_.residual_tol, opts_.relax_factor, logger,
            PrecType(), a, b.values, x.values);

        GKO_CUDA_LAST_IF_ERROR_THROW;
    }

private:
    std::shared_ptr<const CudaExecutor> exec_;
    const BatchRichardsonOptions<remove_complex<value_type>> opts_;
};


template <typename ValueType>
void apply(std::shared_ptr<const CudaExecutor> exec,
           const BatchRichardsonOptions<remove_complex<ValueType>>& opts,
           const BatchLinOp* const a,
           const matrix::BatchDense<ValueType>* const b,
           matrix::BatchDense<ValueType>* const x,
           log::BatchLogData<ValueType>& logdata)
{
    using cu_value_type = cuda_type<ValueType>;
    auto dispatcher = batch_solver::create_dispatcher<ValueType>(
        KernelCaller<cu_value_type>(exec, opts), opts);
    dispatcher.apply(a, nullptr, b, x, logdata);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_RICHARDSON_APPLY_KERNEL);


}  // namespace batch_rich
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
