/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2018

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/gmres_kernels.hpp"


#include "core/base/exception_helpers.hpp"
#include "core/base/math.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
namespace gmres {


template <typename ValueType>
void initialize_1(std::shared_ptr<const CudaExecutor> exec,
                  const matrix::Dense<ValueType> *b,
                  matrix::Dense<ValueType> *b_norm,
                  matrix::Dense<ValueType> *residual,
                  matrix::Dense<ValueType> *givens_sin,
                  matrix::Dense<ValueType> *givens_cos,
                  Array<stopping_status> *stop_status, const int krylov_dim)
{
    NOT_IMPLEMENTED;
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_GMRES_INITIALIZE_1_KERNEL);


template <typename ValueType>
void initialize_2(std::shared_ptr<const CudaExecutor> exec,
                  const matrix::Dense<ValueType> *residual,
                  matrix::Dense<ValueType> *residual_norm,
                  matrix::Dense<ValueType> *residual_norms,
                  matrix::Dense<ValueType> *krylov_bases,
                  Array<size_type> *final_iter_nums, const int krylov_dim)
{
    NOT_IMPLEMENTED;
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_GMRES_INITIALIZE_2_KERNEL);


template <typename ValueType>
void step_1(std::shared_ptr<const CudaExecutor> exec,
            matrix::Dense<ValueType> *next_krylov_basis,
            matrix::Dense<ValueType> *givens_sin,
            matrix::Dense<ValueType> *givens_cos,
            matrix::Dense<ValueType> *residual_norm,
            matrix::Dense<ValueType> *residual_norms,
            matrix::Dense<ValueType> *krylov_bases,
            matrix::Dense<ValueType> *hessenberg_iter,
            const matrix::Dense<ValueType> *b_norm, const size_type iter,
            const Array<stopping_status> *stop_status)
{
    NOT_IMPLEMENTED;
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_GMRES_STEP_1_KERNEL);


template <typename ValueType>
void step_2(std::shared_ptr<const CudaExecutor> exec,
            const matrix::Dense<ValueType> *residual_norms,
            matrix::Dense<ValueType> *krylov_bases,
            matrix::Dense<ValueType> *hessenberg, matrix::Dense<ValueType> *y,
            matrix::Dense<ValueType> *x,
            const Array<size_type> *final_iter_nums,
            const LinOp *preconditioner)
{
    NOT_IMPLEMENTED;
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_GMRES_STEP_2_KERNEL);


}  // namespace gmres
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
