#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/


#include "core/solver/batch_direct_kernels.hpp"


#include "core/matrix/batch_struct.hpp"
#include "cuda/base/cublas_bindings.hpp"
#include "cuda/components/thread_ids.cuh"


namespace gko {
namespace kernels {
namespace cuda {
namespace batch_direct {


constexpr int default_block_size = 256;
#include "common/solver/batch_direct_kernels.hpp.inc"


namespace {

void check_batch(std::shared_ptr<const CudaExecutor> exec, const int nbatch,
                 const int *const info, const bool factorization)
{
    auto host_exec = exec->get_master();
    int *const h_info = host_exec->alloc<int>(nbatch);
    host_exec->copy_from(exec.get(), nbatch, info, h_info);
    for (int i = 0; i < nbatch; i++) {
        if (info[i] < 0 && factorization) {
            std::cerr << "Cublas batch factorization was given an invalid "
                      << "argument at the " << -1 * info[i] << "th position.\n";
        } else if (info[i] < 0 && !factorization) {
            std::cerr << "Cublas batch triangular solve was given an invalid "
                      << "argument at the " << -1 * info[i] << "th position.\n";
        } else if (info[i] > 0 && factorization) {
            std::cerr << "Cublas batch factorization: The " << info[i]
                      << "th matrix was singular.\n";
        }
    }
    host_exec->free(h_info);
}

}  // namespace


template <typename ValueType>
void apply(std::shared_ptr<const CudaExecutor> exec,
           matrix::BatchDense<ValueType> *const a_t,
           matrix::BatchDense<ValueType> *const b_t,
           gko::log::BatchLogData<ValueType> &logdata)
{
    const size_type num_batches = a_t->get_num_batch_entries();
    const int nbatch = static_cast<int>(num_batches);
    const int n = a_t->get_size().at()[0];
    const size_type stride = a_t->get_stride().at();
    const int lda = static_cast<int>(stride);
    const size_type b_stride = b_t->get_stride().at();
    const int nrhs = static_cast<int>(b_t->get_size().at()[0]);
    const int ldb = static_cast<int>(b_stride);

    int *const pivot_array = exec->alloc<int>(nbatch * n);
    int *const info_array = exec->alloc<int>(nbatch);
    ValueType **const matrices = exec->alloc<ValueType *>(nbatch);
    ValueType **const vectors = exec->alloc<ValueType *>(nbatch);
    const int nblk_1 = (nbatch - 1) / default_block_size + 1;
    setup_batch_pointers<<<nblk_1, default_block_size>>>(
        num_batches, n, stride, as_cuda_type(a_t->get_values()),
        as_cuda_type(matrices), nrhs, b_stride, as_cuda_type(b_t->get_values()),
        as_cuda_type(vectors));

    auto handle = cublas::init();
    cublas::batch_getrf(handle, n, matrices, lda, pivot_array, info_array,
                        nbatch);
#ifndef NDEBUG
    check_batch(exec, nbatch, info_array, true);
#endif
    exec->free(info_array);

    int trsm_info{};
    cublas::batch_getrs(handle, HIPBLAS_OP_N, n, nrhs, matrices, lda,
                        pivot_array, vectors, ldb, &trsm_info, nbatch);
    if (trsm_info != 0) {
        std::cerr << "Cublas batch trsm got an illegal param in position "
                  << trsm_info << std::endl;
    }
    cublas::destroy(handle);

    exec->free(matrices);
    exec->free(vectors);
    exec->free(pivot_array);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_DIRECT_APPLY_KERNEL);


template <typename ValueType>
void transpose_scale_copy(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::BatchDense<ValueType> *const scaling_vec,
    const matrix::BatchDense<ValueType> *const orig,
    matrix::BatchDense<ValueType> *const scaled)
{
    const size_type nbatch = orig->get_num_batch_entries();
    const int nrows = static_cast<int>(scaled->get_size().at()[0]);
    const int nrhs = static_cast<int>(scaled->get_size().at()[1]);
    const size_type orig_stride = orig->get_stride().at();
    const size_type scaled_stride = scaled->get_stride().at();
    transpose_scale_copy<<<nbatch, default_block_size>>>(
        nbatch, nrows, nrhs, orig_stride, scaled_stride,
        as_cuda_type(scaling_vec->get_const_values()),
        as_cuda_type(orig->get_const_values()),
        as_cuda_type(scaled->get_values()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_BATCH_DIRECT_TRANSPOSE_SCALE_COPY);


}  // namespace batch_direct
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
