#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/batch_idr_kernels.hpp"


#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/preconditioner/batch_preconditioner_strings.hpp>
#include <ginkgo/core/stop/batch_stop_enum.hpp>

#include "cuda/base/config.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/matrix/batch_struct.hpp"


namespace gko {
namespace kernels {
namespace cuda {


constexpr int default_block_size = 128;
constexpr int sm_multiplier = 4;

/**
 * @brief The batch Idr solver namespace.
 *
 * @ingroup batch_idr
 */
namespace batch_idr {

#include "common/components/uninitialized_array.hpp.inc"


#include "common/log/batch_logger.hpp.inc"
#include "common/matrix/batch_csr_kernels.hpp.inc"
#include "common/matrix/batch_dense_kernels.hpp.inc"
#include "common/preconditioner/batch_identity.hpp.inc"
#include "common/preconditioner/batch_jacobi.hpp.inc"
#include "common/stop/batch_criteria.hpp.inc"


template <typename T>
using BatchIdrOptions = gko::kernels::batch_idr::BatchIdrOptions<T>;

template <typename ValueType>
void apply(std::shared_ptr<const CudaExecutor> exec,
           const BatchIdrOptions<remove_complex<ValueType>> &opts,
           const BatchLinOp *const a,
           const matrix::BatchDense<ValueType> *const left_scale,
           const matrix::BatchDense<ValueType> *const right_scale,
           const matrix::BatchDense<ValueType> *const b,
           matrix::BatchDense<ValueType> *const x,
           log::BatchLogData<ValueType> &logdata) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_IDR_APPLY_KERNEL);


}  // namespace batch_idr
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
