#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2019

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "cuda/components/zero_array.hpp"


namespace gko {
namespace kernels {
namespace cuda {


constexpr int default_block_size = 512;


namespace kernel {


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void zero_array(
    size_type n, ValueType *__restrict__ array)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    if (tidx < n) {
        array[tidx] = zero<ValueType>();
    }
}


}  // namespace kernel


template <typename ValueType>
void zero_array(size_type n, ValueType *array)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(ceildiv(n, block_size.x), 1, 1);
    kernel::zero_array<<<grid_size, block_size, 0, 0>>>(n, array);
}


#define GKO_DECLARE_ZERO_ARRAY(_type) \
    void zero_array<_type>(size_type n, _type * array);
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_ZERO_ARRAY);


}  // namespace cuda
}  // namespace kernels
}  // namespace gko
