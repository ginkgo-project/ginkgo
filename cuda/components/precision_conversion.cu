/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/components/precision_conversion.hpp"


#include "cuda/base/types.hpp"
#include "cuda/components/thread_ids.cuh"


namespace gko {
namespace kernels {
namespace cuda {
namespace components {


constexpr int default_block_size = 512;


#include "common/components/precision_conversion.hpp.inc"


template <typename SourceType, typename TargetType>
void convert_precision(std::shared_ptr<const DefaultExecutor> exec,
                       size_type size, const SourceType *in, TargetType *out)
{
    auto num_blocks = ceildiv(size, default_block_size);
    convert_precision<<<num_blocks, default_block_size>>>(
        size, as_cuda_type(in), as_cuda_type(out));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_CONVERSION(GKO_DECLARE_CONVERT_PRECISION_KERNEL);


}  // namespace components
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
