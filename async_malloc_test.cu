#include <hip/hip_runtime.h>


__global__ void kernel(float* data, int size)
{
#pragma unroll
    for (int i = 0; i < size; i++) {
        data[i] = .5 + i;
    }
}


void free_synchronize()
{
    hipStream_t stream1, stream2;
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    const size_t length = 1 << 10;
    float* dev_a1 = 0;
    float* dev_a2 = 0;
    for (int i = 0; i < 10; i++) {
        hipMalloc(&dev_a1, length);
        kernel<<<1, 1, 0, stream1>>>(dev_a1, length);
        hipMalloc(&dev_a2, length);
        kernel<<<1, 1, 0, stream2>>>(dev_a2, length);
    }
    hipFree(dev_a1);
    hipFree(dev_a2);
}

int main()
{
    free_synchronize();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
