#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2018

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/cgs_kernels.hpp"


#include "core/base/exception_helpers.hpp"
#include "core/base/math.hpp"
#include "gpu/base/math.hpp"
#include "gpu/base/types.hpp"


namespace gko {
namespace kernels {
namespace gpu {
namespace cgs {


constexpr int default_block_size = 512;


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void initialize_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    const ValueType *__restrict__ b, ValueType *__restrict__ r,
    ValueType *__restrict__ r_tld, ValueType *__restrict__ p,
    ValueType *__restrict__ q, ValueType *__restrict__ u,
    ValueType *__restrict__ u_hat, ValueType *__restrict__ v_hat,
    ValueType *__restrict__ t, ValueType *__restrict__ alpha,
    ValueType *__restrict__ beta, ValueType *__restrict__ gamma,
    ValueType *__restrict__ rho_prev, ValueType *__restrict__ rho,
    bool *__restrict__ converged)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;

    if (tidx < num_cols) {
        rho[tidx] = zero<ValueType>();
        alpha[tidx] = one<ValueType>();
        beta[tidx] = one<ValueType>();
        gamma[tidx] = one<ValueType>();
        rho_prev[tidx] = one<ValueType>();
        converged[tidx] = false;
    }

    if (tidx < num_rows * stride) {
        r[tidx] = b[tidx];
        r_tld[tidx] = b[tidx];
        u[tidx] = zero<ValueType>();
        p[tidx] = zero<ValueType>();
        q[tidx] = zero<ValueType>();
        u_hat[tidx] = zero<ValueType>();
        v_hat[tidx] = zero<ValueType>();
        t[tidx] = zero<ValueType>();
    }
}


template <typename ValueType>
void initialize(std::shared_ptr<const GpuExecutor> exec,
                const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *r,
                matrix::Dense<ValueType> *r_tld, matrix::Dense<ValueType> *p,
                matrix::Dense<ValueType> *q, matrix::Dense<ValueType> *u,
                matrix::Dense<ValueType> *u_hat,
                matrix::Dense<ValueType> *v_hat, matrix::Dense<ValueType> *t,
                matrix::Dense<ValueType> *alpha, matrix::Dense<ValueType> *beta,
                matrix::Dense<ValueType> *gamma,
                matrix::Dense<ValueType> *rho_prev,
                matrix::Dense<ValueType> *rho, Array<bool> *converged)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(b->get_size().num_rows * b->get_stride(), block_size.x), 1, 1);

    initialize_kernel<<<grid_size, block_size, 0, 0>>>(
        b->get_size().num_rows, b->get_size().num_cols, b->get_stride(),
        as_cuda_type(b->get_const_values()), as_cuda_type(r->get_values()),
        as_cuda_type(r_tld->get_values()), as_cuda_type(p->get_values()),
        as_cuda_type(q->get_values()), as_cuda_type(u->get_values()),
        as_cuda_type(u_hat->get_values()), as_cuda_type(v_hat->get_values()),
        as_cuda_type(t->get_values()), as_cuda_type(alpha->get_values()),
        as_cuda_type(beta->get_values()), as_cuda_type(gamma->get_values()),
        as_cuda_type(rho_prev->get_values()), as_cuda_type(rho->get_values()),
        as_cuda_type(converged->get_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_INITIALIZE_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void test_convergence_kernel(
    size_type num_cols, remove_complex<ValueType> rel_residual_goal,
    const ValueType *__restrict__ tau, const ValueType *__restrict__ orig_tau,
    bool *__restrict__ converged, bool *__restrict__ all_converged)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    if (tidx < num_cols) {
        if (abs(tau[tidx]) < rel_residual_goal * abs(orig_tau[tidx])) {
            converged[tidx] = true;
        }
        // because only false is written to all_converged, write conflicts
        // should not cause any problem
        else if (converged[tidx] == false) {
            *all_converged = false;
        }
    }
}

template <typename ValueType>
void test_convergence(std::shared_ptr<const GpuExecutor> exec,
                      const matrix::Dense<ValueType> *tau,
                      const matrix::Dense<ValueType> *orig_tau,
                      remove_complex<ValueType> rel_residual_goal,
                      Array<bool> *converged, bool *all_converged)
{
    Array<bool> d_all_converged(exec, 1);
    Array<bool> all_converged_array(exec->get_master());

    // initialize all_converged with true
    *all_converged = true;
    all_converged_array.manage(1, all_converged);
    d_all_converged = all_converged_array;

    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(ceildiv(tau->get_size().num_cols, block_size.x), 1, 1);

    test_convergence_kernel<<<grid_size, block_size, 0, 0>>>(
        tau->get_size().num_cols, rel_residual_goal,
        as_cuda_type(tau->get_const_values()),
        as_cuda_type(orig_tau->get_const_values()),
        as_cuda_type(converged->get_data()),
        as_cuda_type(d_all_converged.get_data()));

    all_converged_array = d_all_converged;
    all_converged_array.release();
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_TEST_CONVERGENCE_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void step_1_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    const ValueType *__restrict__ r, ValueType *__restrict__ u,
    ValueType *__restrict__ p, const ValueType *__restrict__ q,
    ValueType *__restrict__ beta, const ValueType *__restrict__ rho,
    const ValueType *__restrict__ rho_prev, const bool *__restrict__ converged)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto col = tidx % stride;

    if (col >= num_cols || tidx >= num_rows * stride || converged[col]) {
        return;
    }
    if (rho_prev[col] != zero<ValueType>()) {
        beta[col] = rho[col] / rho_prev[col];
        u[tidx] = r[tidx] + beta[col] * q[tidx];
        p[tidx] = u[tidx] + beta[col] * (q[tidx] + beta[col] * p[tidx]);
    }
}


template <typename ValueType>
void step_1(std::shared_ptr<const GpuExecutor> exec,
            const matrix::Dense<ValueType> *r, matrix::Dense<ValueType> *u,
            matrix::Dense<ValueType> *p, const matrix::Dense<ValueType> *q,
            matrix::Dense<ValueType> *beta, const matrix::Dense<ValueType> *rho,
            const matrix::Dense<ValueType> *rho_prev,
            const Array<bool> &converged)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(p->get_size().num_rows * p->get_stride(), block_size.x), 1, 1);

    step_1_kernel<<<grid_size, block_size, 0, 0>>>(
        p->get_size().num_rows, p->get_size().num_cols, p->get_stride(),
        as_cuda_type(r->get_const_values()), as_cuda_type(u->get_values()),
        as_cuda_type(p->get_values()), as_cuda_type(q->get_const_values()),
        as_cuda_type(beta->get_values()), as_cuda_type(rho->get_const_values()),
        as_cuda_type(rho_prev->get_const_values()),
        as_cuda_type(converged.get_const_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_STEP_1_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void step_2_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    const ValueType *__restrict__ u, const ValueType *__restrict__ v_hat,
    ValueType *__restrict__ q, ValueType *__restrict__ t,
    ValueType *__restrict__ alpha, const ValueType *__restrict__ rho,
    const ValueType *__restrict__ gamma, const bool *__restrict__ converged)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto col = tidx % stride;

    if (col >= num_cols || tidx >= num_rows * stride || converged[col]) {
        return;
    }
    if (gamma[col] != zero<ValueType>()) {
        alpha[col] = rho[col] / gamma[col];
        q[tidx] = u[tidx] - alpha[col] * v_hat[tidx];
        t[tidx] = u[tidx] + q[tidx];
    }
}


template <typename ValueType>
void step_2(std::shared_ptr<const GpuExecutor> exec,
            const matrix::Dense<ValueType> *u,
            const matrix::Dense<ValueType> *v_hat, matrix::Dense<ValueType> *q,
            matrix::Dense<ValueType> *t, matrix::Dense<ValueType> *alpha,
            const matrix::Dense<ValueType> *rho,
            const matrix::Dense<ValueType> *gamma, const Array<bool> &converged)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(u->get_size().num_rows * u->get_stride(), block_size.x), 1, 1);

    step_2_kernel<<<grid_size, block_size, 0, 0>>>(
        u->get_size().num_rows, u->get_size().num_cols, u->get_stride(),
        as_cuda_type(u->get_const_values()),
        as_cuda_type(v_hat->get_const_values()), as_cuda_type(q->get_values()),
        as_cuda_type(t->get_values()), as_cuda_type(alpha->get_values()),
        as_cuda_type(rho->get_const_values()),
        as_cuda_type(gamma->get_const_values()),
        as_cuda_type(converged.get_const_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_STEP_2_KERNEL);


template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void step_3_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    size_type x_stride, const ValueType *__restrict__ t,
    const ValueType *__restrict__ v_hat, ValueType *__restrict__ r,
    ValueType *__restrict__ x, const ValueType *__restrict__ alpha,
    const bool *__restrict__ converged)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto row = tidx / stride;
    const auto col = tidx % stride;
    if (col >= num_cols || tidx >= num_rows * stride || converged[col]) {
        return;
    }
    const auto x_pos = row * x_stride + col;
    auto t_x = x[x_pos] + alpha[col] * v_hat[tidx];
    auto t_r = r[tidx] - alpha[col] * t[tidx];
    x[x_pos] = t_x;
    r[tidx] = t_r;
}


template <typename ValueType>
void step_3(std::shared_ptr<const GpuExecutor> exec,
            const matrix::Dense<ValueType> *t,
            const matrix::Dense<ValueType> *u_hat, matrix::Dense<ValueType> *r,
            matrix::Dense<ValueType> *x, const matrix::Dense<ValueType> *alpha,
            const Array<bool> &converged)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(t->get_size().num_rows * t->get_stride(), block_size.x), 1, 1);

    step_3_kernel<<<grid_size, block_size, 0, 0>>>(
        t->get_size().num_rows, t->get_size().num_cols, t->get_stride(),
        x->get_stride(), as_cuda_type(t->get_const_values()),
        as_cuda_type(u_hat->get_const_values()), as_cuda_type(r->get_values()),
        as_cuda_type(x->get_values()), as_cuda_type(alpha->get_const_values()),
        as_cuda_type(converged.get_const_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_STEP_3_KERNEL);


}  // namespace cgs
}  // namespace gpu
}  // namespace kernels
}  // namespace gko
